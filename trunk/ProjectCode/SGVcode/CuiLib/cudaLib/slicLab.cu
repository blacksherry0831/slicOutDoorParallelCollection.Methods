#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include "math.h"
/*------------------------------------------------------------------------------------------*/
#include <stdio.h>
#include <assert.h>
#include <float.h>
#include <limits.h>
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaInit(void);
extern "C" bool cudaDeInit(void);
extern "C" void cudaGetLastError_Sync(void);
/*------------------------------------------------------------------------------------------*/
extern "C" void Fill_Matrix_Int(int* _dev_m, const int _sz, const int _v);
extern "C" void Fill_Matrix_Float(float* _dev_m, const int _sz, const float _v);
extern "C" void Fill_Matrix_Db(double* _dev_m, const int _sz, const double _v);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void PerformSuperpixelSLIC_gpu_lab(
	const int			_kseedsSize,
	const int			_width,
	const int			_height,
	const double*		_host_matrix_l,
	const double*		_host_matrix_a,
	const double*		_host_matrix_b,
	const double*		_host_matrix_x,
	const double*		_host_matrix_y,
	double*				_host_kseedsl,
	double*				_host_kseedsa,
	double*				_host_kseedsb,
	double*				_host_kseedsx,
	double*				_host_kseedsy,
	int*				_host_klabels,
	const double		_invwt);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
__inline__  __device__  double CalDistLAB_Basic_Square(
	const double	L0,
	const double	A0,
	const double	B0,
	const double	X0,
	const double	Y0,
	const double	L1,
	const double	A1,
	const double	B1,
	const double	X1,
	const double	Y1,
	const double	INVWT)
{
	const double dst = (L1 - L0)*(L1 - L0) + (B1 - B0)*(B1 - B0) + (A1 - A0)*(A1 - A0);
	const double dst_xy = (X1 - X0)*(X1 - X0) + (Y1 - Y0)*(Y1 - Y0);
	const double dist_labxy = dst + INVWT * dst_xy;
	return dist_labxy;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void Statistics_Lab_Seed_collect_CPU_Incompletion_host(
	const int			_kseedsSize,
	const int			_width,
	const int			_height,

	const double*		_h_l,
	const double*		_h_a,
	const double*		_h_b,
	const double*		_h_x,
	const double*		_h_y,

	const int*			_h_klabels,

	double*				_h_kseed_L,
	double*				_h_kseed_A,
	double*				_h_kseed_B,
	double*				_h_kseed_X,
	double*				_h_kseed_Y,
	double*				_h_kseed_SZ)
{
	const int ZERO = 0;
	const int IMG_SZ = _width*_height;
	const int SEED_BUFF_SZ = _kseedsSize * sizeof(double);
	const int IMG_BUFF_SZ_DB = IMG_SZ * sizeof(double);
	const int IMG_BUFF_SZ_INT = IMG_SZ * sizeof(int);
	const int IMG_BUFF_SZ_UINT = IMG_SZ * sizeof(unsigned int);
	
	memset(_h_kseed_L,				ZERO,	SEED_BUFF_SZ);
	memset(_h_kseed_A,				ZERO,	SEED_BUFF_SZ);
	memset(_h_kseed_B,				ZERO,	SEED_BUFF_SZ);
	memset(_h_kseed_X,				ZERO,	SEED_BUFF_SZ);
	memset(_h_kseed_Y,				ZERO,	SEED_BUFF_SZ);
	memset(_h_kseed_SZ,				ZERO,	SEED_BUFF_SZ);
	
		int ind(0);
		for (register int r = 0; r < _height; r++){
			for (register int c = 0; c < _width; c++){	
				const int label_i = _h_klabels[ind];
						_h_kseed_L[label_i]		+= _h_l[ind];
						_h_kseed_A[label_i]		+= _h_a[ind];
						_h_kseed_B[label_i]		+= _h_b[ind];
						_h_kseed_X[label_i]		+= c;
						_h_kseed_Y[label_i]		+= r;
						_h_kseed_SZ[label_i]	+= 1.0;
						ind++;
			}
		}
			
		for (int k = 0; k < _kseedsSize; k++)
		{
			if (_h_kseed_SZ[k] <= 0) 
				_h_kseed_SZ[k] = 1;

			const double inv_k = 1.0 / _h_kseed_SZ[k];
			//computing inverse now to multiply, than divide later		

			_h_kseed_L[k] *= inv_k;
			_h_kseed_A[k] *= inv_k;
			_h_kseed_B[k] *= inv_k;
			_h_kseed_X[k] *= inv_k;
			_h_kseed_Y[k] *= inv_k;
			
		}
	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void Statistics_Lab_Seed_collect_CPU_Incompletion(
	const int			_kseedsSize,
	const int			_width,
	const int			_height,
	const double*		_h_l,
	const double*		_h_a,
	const double*		_h_b,
	const double*		_h_x,
	const double*		_h_y,
	const int*			_dev_klabels,
	double*				_dev_kseed_L,
	double*				_dev_kseed_A,
	double*				_dev_kseed_B,
	double*				_dev_kseed_X,
	double*				_dev_kseed_Y)
{

	const int ZERO = 0;
	const int IMG_SZ = _width*_height;
	const int SEED_BUFF_SZ = _kseedsSize * sizeof(double);
	const int IMG_BUFF_SZ_DB = IMG_SZ * sizeof(double);
	const int IMG_BUFF_SZ_INT = IMG_SZ * sizeof(int);
	const int IMG_BUFF_SZ_UINT = IMG_SZ * sizeof(unsigned int);

	int*				h_klabels =		new int[IMG_SZ];
	double*				h_kseed_L =		new double[_kseedsSize];
	double*				h_kseed_A =		new double[_kseedsSize];
	double*				h_kseed_B =		new double[_kseedsSize];
	double*				h_kseed_X =		new double[_kseedsSize];
	double*				h_kseed_Y =		new double[_kseedsSize];
	double*				h_kseed_SZ =	new double[_kseedsSize];

	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(h_klabels, _dev_klabels, IMG_BUFF_SZ_INT, hipMemcpyDeviceToHost);

	Statistics_Lab_Seed_collect_CPU_Incompletion_host(
		_kseedsSize,
		_width,
		_height,
		_h_l,
		_h_a,
		_h_b,
		_h_x,
		_h_y,
		h_klabels,
		h_kseed_L,
		h_kseed_A,
		h_kseed_B,
		h_kseed_X,
		h_kseed_Y,
		h_kseed_SZ);

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	cudaStatus = hipMemcpy(_dev_kseed_L,		h_kseed_L, SEED_BUFF_SZ, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(_dev_kseed_A,		h_kseed_A, SEED_BUFF_SZ, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(_dev_kseed_B,		h_kseed_B, SEED_BUFF_SZ, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(_dev_kseed_X,		h_kseed_X, SEED_BUFF_SZ, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(_dev_kseed_Y,		h_kseed_Y, SEED_BUFF_SZ, hipMemcpyHostToDevice);
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	delete[] 		h_klabels;
	delete[] 		h_kseed_L;
	delete[] 		h_kseed_A;
	delete[] 		h_kseed_B;
	delete[] 		h_kseed_X;
	delete[] 		h_kseed_Y;
	delete[] 		h_kseed_SZ;

}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
__inline__ __device__  void SetPixelLablesLab(
	const int* x_offset,
	const int* y_offset,
	const int offsetSize,
	
	const int SeedWH,
	const int labelsX,
	const int labelsY,
	const int x,
	const int y,
	const int m_width,
	const int m_height,
	const double* dev_kseeds_L,
	const double* dev_kseeds_A,
	const double* dev_kseeds_B,
	const double* dev_kseeds_X,
	const double* dev_kseeds_Y,
	const double* dev_L,
	const double* dev_A,
	const double* dev_B,
	const double* dev_X,
	const double* dev_Y,
	int*   klabels,
	double* distvec,
	const double _invwt)
{
		double dist = 0;
		//////////////////////////////////////////////
		const int i = y*m_width + x;
		////////////////////////////////////////////
		for (int xyi = 0; xyi<offsetSize; xyi++) {
			const int nx = labelsX + x_offset[xyi];
			const int ny = labelsY + y_offset[xyi];
			if (nx >= 0 && nx<SeedWH&&ny >= 0 && ny<SeedWH) {
				const int ni = nx + ny*SeedWH;
				{
					 dist = CalDistLAB_Basic_Square(
													dev_L[i], dev_A[i], dev_B[i], dev_X[i], dev_Y[i],
													dev_kseeds_L[ni], dev_kseeds_A[ni], dev_kseeds_B[ni], dev_kseeds_X[ni], dev_kseeds_Y[ni],
													_invwt);

					if (dist < distvec[i])
					{
						distvec[i] = dist;
						klabels[i] = ni;
					}

				}
			}

		}

}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnLab_9Seeds(
	const int kseedsSize,
		
	const int _width,
	const int _height,
	
	const double*		dev_L,
	const double*		dev_A,
	const double*		dev_B,
	const double*		dev_X,
	const double*		dev_Y,

	const double*	dev_kseed_L,
	const double*	dev_kseed_A,
	const double*	dev_kseed_B,
	const double*	dev_kseed_X,
	const double*	dev_kseed_Y,

	int*   klabels,
	double* distvec,

	const double _invwt)
{
	const int x_offset[] = { 0,1,-1,0,0,1,-1,1,-1 };
	const int y_offset[] = { 0,0,0,1,-1,1,-1,-1,1 };

	{
		const int x = blockIdx.x*blockDim.x + threadIdx.x;
		const int y = blockIdx.y*blockDim.y + threadIdx.y;
		const int SeedWH = sqrt((float)kseedsSize);

		const int offsetSize = sizeof(x_offset) / sizeof(int);
		
		if (x >= 0 && x<_width&&y >= 0 && y<_height) {
			const int ind = x + y*_width;
			const int labels_ind = klabels[ind];
			assert(labels_ind >= -1 && labels_ind<kseedsSize);

			int labelsX = 0;
			int labelsY = 0;

			if (labels_ind == -1) {
					const float x_step = 1.0*_width		/	SeedWH;
					const float y_step = 1.0*_height	/	SeedWH;
					labelsX = x / x_step;
					labelsY = y / y_step;				
			}else {
					labelsX = labels_ind	%	SeedWH;
					labelsY = labels_ind	/	SeedWH;
			}

			SetPixelLablesLab(x_offset, y_offset, offsetSize,
					SeedWH, labelsX, labelsY, x, y,
					_width, _height,
					dev_kseed_L, dev_kseed_A, dev_kseed_B, dev_kseed_X, dev_kseed_Y,
					dev_L, dev_A, dev_B, dev_X, dev_Y,
					klabels,
					distvec,
					_invwt);

		}

	}

	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void SetMinimumOnLab_9Seeds_Super(
	const int kseedsSize,

	const int m_width,
	const int m_height,

	const double* dev_L,
	const double* dev_A,
	const double* dev_B,
	const double* dev_X,
	const double* dev_Y,

	double*	dev_kseed_L,
	double*	dev_kseed_A,
	double*	dev_kseed_B,
	double*	dev_kseed_X,
	double*	dev_kseed_Y,

	int*   klabels,
	double* distvec,

	const double _invwt)
{
	hipEvent_t start = 0, stop = 0;
	float costtime_ms=0, costtime_us = 0;
	hipError_t cudaStatus_t = hipEventRecord(start, 0);
	{
		dim3    threadsPerBlock(16, 16);
		dim3    numBlock((m_width + threadsPerBlock.x - 1) / threadsPerBlock.x, (m_height + threadsPerBlock.y - 1) / threadsPerBlock.y);

		SetMinimumOnLab_9Seeds <<<numBlock, threadsPerBlock >>>(
			kseedsSize,
			m_width,
			m_height,

			dev_L,
			dev_A,
			dev_B,
			dev_X,
			dev_Y,

			dev_kseed_L,
			dev_kseed_A,
			dev_kseed_B,
			dev_kseed_X,
			dev_kseed_Y,

			klabels,
			distvec,

			_invwt);

		cudaGetLastError_Sync();
	
	}
	cudaStatus_t = hipEventRecord(stop, 0);
	cudaGetLastError_Sync();
	cudaStatus_t = hipEventElapsedTime(&costtime_ms, start, stop);
	costtime_us = costtime_ms * 1000;
	hipEventDestroy(start);
	hipEventDestroy(stop);

}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_gpu_lab_dev(
	const int _kseedsSize,
	const int _width,
	const int _height,
	
	const double*		dev_L,
	const double*		dev_A,
	const double*		dev_B,
	const double*		dev_X,
	const double*		dev_Y,
	
	double*				_dev_kseedsl,
	double*				_dev_kseedsa,
	double*				_dev_kseedsb,
	double*				_dev_kseedsx,
	double*				_dev_kseedsy,

	int*				_dev_klabels,
	double*				_dev_distvec,
	const double		_invwt)
{
		const int SZ = _width*_height;
		
		Fill_Matrix_Db(_dev_distvec, SZ , DBL_MAX);
	
		SetMinimumOnLab_9Seeds_Super(
			_kseedsSize,
			_width,
			_height,

			dev_L,
			dev_A,
			dev_B,
			dev_X,
			dev_Y,

			_dev_kseedsl,
			_dev_kseedsa,
			_dev_kseedsb,
			_dev_kseedsx,
			_dev_kseedsy,

			_dev_klabels,
			_dev_distvec,
			_invwt);
		
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/

/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_gpu_lab(
	const int			_kseedsSize,
	const int			_width,
	const int			_height,
	const double*		_host_matrix_l,
	const double*		_host_matrix_a,
	const double*		_host_matrix_b,
	const double*		_host_matrix_x,
	const double*		_host_matrix_y,
	double*				_host_kseedsl,
	double*				_host_kseedsa,
	double*				_host_kseedsb,
	double*				_host_kseedsx,
	double*				_host_kseedsy,
	int*				_host_klabels,
	const double		_invwt)
{
	bool cuda_stat_t = cudaInit();
	assert(cuda_stat_t == true);
		{
		const int IMG_SZ = _width*_height;
		const int SEED_BUFF_SZ = _kseedsSize * sizeof(double);
		const int IMG_BUFF_SZ_DB = IMG_SZ * sizeof(double);
		const int IMG_BUFF_SZ_INT = IMG_SZ * sizeof(int);
		
					hipError_t cudaStatus;

					double*				dev_kseedsl = nullptr;
					double*				dev_kseedsa = nullptr;
					double*				dev_kseedsb = nullptr;
					double*				dev_kseedsx = nullptr;
					double*				dev_kseedsy = nullptr;

					int*				dev_klabels = nullptr;

					double*				dev_matrix_l = nullptr;
					double*				dev_matrix_a = nullptr;
					double*				dev_matrix_b = nullptr;
					double*				dev_matrix_x = nullptr;
					double*				dev_matrix_y = nullptr;

					double*				dev_distvec = nullptr;
		
					cudaStatus = hipMalloc((void**)&dev_kseedsl, SEED_BUFF_SZ);
					cudaStatus = hipMalloc((void**)&dev_kseedsa, SEED_BUFF_SZ);
					cudaStatus = hipMalloc((void**)&dev_kseedsb, SEED_BUFF_SZ);
					cudaStatus = hipMalloc((void**)&dev_kseedsx, SEED_BUFF_SZ);
					cudaStatus = hipMalloc((void**)&dev_kseedsy, SEED_BUFF_SZ);
					cudaStatus = hipMalloc((void**)&dev_klabels, IMG_BUFF_SZ_INT);
					cudaStatus = hipMalloc((void**)&dev_matrix_l,  IMG_BUFF_SZ_DB);
					cudaStatus = hipMalloc((void**)&dev_matrix_a,  IMG_BUFF_SZ_DB);
					cudaStatus = hipMalloc((void**)&dev_matrix_b,  IMG_BUFF_SZ_DB);
					cudaStatus = hipMalloc((void**)&dev_matrix_x,	IMG_BUFF_SZ_DB);
					cudaStatus = hipMalloc((void**)&dev_matrix_y,	IMG_BUFF_SZ_DB);

					cudaStatus = hipMalloc((void**)&dev_distvec,	IMG_BUFF_SZ_DB);
						{
							cudaStatus = hipMemcpy(dev_kseedsl,	_host_kseedsl,	SEED_BUFF_SZ,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_kseedsa,	_host_kseedsa,	SEED_BUFF_SZ,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_kseedsb,	_host_kseedsb,	SEED_BUFF_SZ,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_kseedsx,	_host_kseedsx,	SEED_BUFF_SZ,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_kseedsy,	_host_kseedsy,	SEED_BUFF_SZ,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_klabels,	_host_klabels,	IMG_BUFF_SZ_INT,hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_matrix_l,	_host_matrix_l,	IMG_BUFF_SZ_DB,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_matrix_a,	_host_matrix_a,	IMG_BUFF_SZ_DB,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_matrix_b,	_host_matrix_b,	IMG_BUFF_SZ_DB,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_matrix_x,	_host_matrix_x,	IMG_BUFF_SZ_DB, hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_matrix_y,	_host_matrix_y,	IMG_BUFF_SZ_DB, hipMemcpyHostToDevice);
														
							cudaGetLastError_Sync();
								
							const int ITR_TIMES = 2;

							for (int itr = 0; itr < ITR_TIMES; itr++)
							{
									PerformSuperpixelSLIC_gpu_lab_dev(
									_kseedsSize,
									_width,
									_height,

									dev_matrix_l,
									dev_matrix_a,
									dev_matrix_b,
									dev_matrix_x,
									dev_matrix_y,

									dev_kseedsl,
									dev_kseedsa,
									dev_kseedsb,
									dev_kseedsx,
									dev_kseedsy,

									dev_klabels,
									dev_distvec,
									_invwt);

									Statistics_Lab_Seed_collect_CPU_Incompletion(
										_kseedsSize,
										_width,
										_height,


										_host_matrix_l,
										_host_matrix_a,
										_host_matrix_b,
										_host_matrix_x,
										_host_matrix_y,


										dev_klabels,
										dev_kseedsl,
										dev_kseedsa,
										dev_kseedsb,
										dev_kseedsx,
										dev_kseedsy);
							}
							

							cudaStatus = hipMemcpy(_host_kseedsl, dev_kseedsl, SEED_BUFF_SZ, hipMemcpyDeviceToHost);
							cudaStatus = hipMemcpy(_host_kseedsa, dev_kseedsa, SEED_BUFF_SZ, hipMemcpyDeviceToHost);
							cudaStatus = hipMemcpy(_host_kseedsb, dev_kseedsb, SEED_BUFF_SZ, hipMemcpyDeviceToHost);
							cudaStatus = hipMemcpy(_host_kseedsx, dev_kseedsx, SEED_BUFF_SZ, hipMemcpyDeviceToHost);
							cudaStatus = hipMemcpy(_host_kseedsy, dev_kseedsy, SEED_BUFF_SZ, hipMemcpyDeviceToHost);
							cudaStatus = hipMemcpy(_host_klabels, dev_klabels, IMG_BUFF_SZ_INT, hipMemcpyDeviceToHost);

						}
					hipFree(dev_kseedsl);
					hipFree(dev_kseedsa);
					hipFree(dev_kseedsb);
					hipFree(dev_kseedsx);
					hipFree(dev_kseedsy);
					hipFree(dev_klabels);
					hipFree(dev_matrix_l);
					hipFree(dev_matrix_a);
					hipFree(dev_matrix_b);
					hipFree(dev_matrix_x);
					hipFree(dev_matrix_y);
					hipFree(dev_distvec);

		}	
	cuda_stat_t = cudaDeInit();
	assert(cuda_stat_t == true);
}

/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/