#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
/*------------------------------------------------------------------------------------------*/
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
/*------------------------------------------------------------------------------------------*/
#include <stdio.h>
#include <assert.h>
#include <iostream>
/*------------------------------------------------------------------------------------------*/
#if linux||__linux||__linux__ || __GNUC__

#include <unistd.h>    
#include <sys/types.h>  
#include <sys/syscall.h>//Linux system call for thread id
#include <pthread.h>

#endif
/*------------------------------------------------------------------------------------------*/
#ifdef _MSC_VER

#include <windows.h>

#endif
/*------------------------------------------------------------------------------------------*/
#define TIME_COST_FUNC (1)
/*------------------------------------------------------------------------------------------*/
extern int  GetTimeCostFunc();
extern void SetTimeCostFunc(int _t);
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaInit(void);
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaDeInit(void);
/*------------------------------------------------------------------------------------------*/
extern "C" void cudaGetLastError_Sync(void);
/*------------------------------------------------------------------------------------------*/
extern "C" void init_Matrix_Db(double* _dev_m, const int _w, const int _h, const double _d);
extern "C" void init_Matrix_Int(int* _dev_m, const int _w, const int _h, const int _v);
extern "C" void init_Matrix_UL(unsigned long * _dev_m, const int _w, const int _h, const unsigned long _d);
/*------------------------------------------------------------------------------------------*/
extern "C" void Fill_Matrix_Int(int* _dev_m, const int _sz, const int _v);
extern "C" void Fill_Matrix_Float(float* _dev_m, const int _sz, const float _v);
extern "C" void Fill_Matrix_Db(double* _dev_m, const int _sz, const double _v);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
int mTimeCostFunc = TIME_COST_FUNC;
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
int GetTimeCostFunc()
{
	return mTimeCostFunc;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
void SetTimeCostFunc(int _t)
{
	mTimeCostFunc=_t;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
int GetThreadIdSelfwinlinux(void)
{

	int thread_id = 0;
#if _DEBUG
	int device_count;
	hipGetDeviceCount(&device_count);
#endif


#if _MSC_VER
	thread_id = ::GetCurrentThreadId();
#elif linux||__linux||__linux__||__GNUC__
	//thread_id=(struct pthread_fake *)pthread_self()->tid;
	thread_id = syscall(__NR_gettid);
#else
	thread_id = -1;
	assert(0);
#endif

	return thread_id;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*cuda��ʼ��
*@retval true ��ʼ���ɹ�
*@retval false ��ʼ��ʧ��
*/
/*------------------------------------------------------------------------------------------*/
bool cudaInit(void)
{
		 int i;
		 int device_count;

		 if( hipGetDeviceCount(&device_count) )
		 {
			 printf(" There is zero device beyond 1.0\n"); 
			 return false;
		 }
#if _DEBUG
		//printf("CUDA DEVICE NUMS: %d \n",device_count);
#endif
		 // �ҵ�һ�����õ��豸
		 for(i=0;i<device_count;i++)
		 {
			  struct hipDeviceProp_t device_prop;
			  if(hipGetDeviceProperties(&device_prop,i)==hipSuccess)
			  {
#if 0
	printf("device properties is :\n"
			"\t device name is %s\n"
			"\t totalGlobalMem is %d\n"
			"\t sharedMemPerBlock is %d\n"
			"\t regsPerBlock is %d\n"
			"\t warpSize is %d\n"
			"\t memPitch is %d\n"
			"\t maxThreadsPerBlock is %d\n"
			"\t maxThreadsDim [3] is %d X %d X %d\n"
			"\t maxGridSize [3] is %d X %d X %d\n"
			"\t totalConstMem is %d\n"
			"\t device version is major %d ,minor %d\n"
			"\t clockRate is %d\n"
			"\t textureAlignment is %d\n"
			"\t deviceOverlap is %d\n"
			"\t multiProcessorCount is %d\n",
			device_prop.name,
			device_prop.totalGlobalMem,
			device_prop.sharedMemPerBlock,
			device_prop.regsPerBlock,
			device_prop.warpSize,
			device_prop.memPitch,
			device_prop.maxThreadsPerBlock,
			device_prop.maxThreadsDim[0],device_prop.maxThreadsDim[1],device_prop.maxThreadsDim[2],
			device_prop.maxGridSize[0],device_prop.maxGridSize[1],device_prop.maxGridSize[2],
			device_prop.totalConstMem,
			device_prop.major,device_prop.minor,
			device_prop.clockRate,
			device_prop.textureAlignment,
			device_prop.deviceOverlap,
			device_prop.multiProcessorCount);
#endif				
					/* if(hipSetDevice(i)==hipSuccess){
						 printf("USE GPU ID: %d \n",i);
						 return true;
					 }*/
				break;
			  }

		 }
		 
		 if(i==device_count)
		 {
			  printf("Get the propertites of device occurred error\n");
			  return false;
		 }
		 //�п����豸
		 int default_id,expect_id;		 
		 int thread_id=GetThreadIdSelfwinlinux();
		 expect_id=thread_id%device_count;
		
		 printf("CUDA NUMS: %d, ID: %d,EXPECT: %d \n",device_count,thread_id,expect_id);

		 hipGetDevice(&default_id);

		 if(default_id==expect_id){
			 return true;
		 }else{			 
			 //�豸����
			 if(hipSetDevice(expect_id)==hipSuccess){
				 //printf("USE GPU ID: %d \n",expect_id);
				 return true;
			 }
		 }
		 
		 //////////////////////////////////////////
	//	 int ket=kernelExecTimeoutEnabled();
		 /////////////////////////////////////////
		 return false;
	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*cudaȥ��ʼ��
*@return 
*@retval true ��ʼ���ɹ�
*@retval false ��ʼ��ʧ��
*/
/*------------------------------------------------------------------------------------------*/
bool cudaDeInit(void)
{
#if 0
	hipError_t cudaStatus;
	cudaStatus=hipDeviceReset();
	if(cudaStatus!=hipSuccess){
		return false;
	}else{
		return true;
	}
#else
	return true;
#endif
	
}
/*------------------------------------------------------------------------------------------*/
/**
*Cuda�ȴ���ʱ��ȡ�������
*
*
*/
/*------------------------------------------------------------------------------------------*/
void cudaGetLastError_Sync(void)
{
	hipError_t cudaStatus_syn,cudaStatus;
	const char  *errstr=NULL;
	
	do 
	{

		cudaStatus_syn=hipDeviceSynchronize();
		if (cudaStatus_syn==hipSuccess)
		{
			break;

		}else{
			cudaStatus=hipGetLastError();
			if (cudaStatus!=hipSuccess){
				errstr=hipGetErrorString(cudaStatus);
				printf("hipDeviceSynchronize: %s",errstr);
			}
		}
		
	} while (cudaStatus_syn!=hipSuccess);

}
/*------------------------------------------------------------------------------------------*/
/**
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMatrixUnsignedLong(unsigned long *matrix, const int width, const int height, const unsigned long value_t)
{
	int ind = blockIdx.x*blockDim.x + threadIdx.x;
	const int offset = blockDim.x*gridDim.x;
	const int size_t = width*height;

	while (ind<size_t) {
		matrix[ind] = value_t;
		ind += offset;
	}

}
/*------------------------------------------------------------------------------------------*/
/**
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMatrixDouble(double *matrix,const int width,const int height,const double value_t)
{
	int ind = blockIdx.x*blockDim.x + threadIdx.x;
	const int offset = blockDim.x*gridDim.x;
	const int size_t = width*height;
	/////////////////////////////////////////
	while (ind<size_t) {
		matrix[ind] = value_t;
		ind += offset;
	}
	/////////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMatrixInt(int* _matrix, const int width, const int height, const int _v)
{
	int ind = blockIdx.x*blockDim.x + threadIdx.x;
	const int offset = blockDim.x*gridDim.x;
	const int size_t = width*height;
	while (ind<size_t) {
		_matrix[ind] = _v;
		ind += offset;
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void init_Matrix_UL(unsigned long * _dev_m, const int _w, const int _h, const unsigned long _d)
{
	hipEvent_t start = 0, stop = 0;
	float costtime_ms = 0, costtime_us = 0;
	hipError_t cudaStatus_t = hipEventRecord(start, 0);
	{
		SetMatrixUnsignedLong << <16, 16 * 16 >> >(_dev_m, _w, _h, _d);
		cudaGetLastError_Sync();
	}
	cudaStatus_t = hipEventRecord(stop, 0);
	cudaGetLastError_Sync();
	cudaStatus_t = hipEventElapsedTime(&costtime_ms, start, stop);
	costtime_us = costtime_ms * 1000;
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void init_Matrix_Db(double* _dev_m, const int _w, const int _h, const double _d)
{
	hipEvent_t start = 0, stop = 0;
	float costtime_ms = 0, costtime_us = 0;
	hipError_t cudaStatus_t = hipEventRecord(start, 0);
	{
		SetMatrixDouble <<<16, 16 * 16 >>>(_dev_m, _w, _h, _d);
		cudaGetLastError_Sync();
	}
	cudaStatus_t = hipEventRecord(stop, 0);
	cudaGetLastError_Sync();
	cudaStatus_t = hipEventElapsedTime(&costtime_ms, start, stop);
	costtime_us = costtime_ms * 1000;
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void init_Matrix_Int(int* _dev_m, const int _w, const int _h, const int _v)
{
	hipEvent_t start = 0, stop = 0;
	float costtime_ms = 0, costtime_us = 0;
	hipError_t cudaStatus_t = hipEventRecord(start, 0);
	{
		SetMatrixInt <<<16, 16 * 16 >>>(_dev_m, _w, _h, _v);
		cudaGetLastError_Sync();
	}
	cudaStatus_t = hipEventRecord(stop, 0);
	cudaGetLastError_Sync();
	cudaStatus_t = hipEventElapsedTime(&costtime_ms, start, stop);
	costtime_us = costtime_ms * 1000;
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*/
/*------------------------------------------------------------------------------------------*/
void GetCudaDevProp(int _d)
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int blocks = prop.multiProcessorCount;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*/
/*------------------------------------------------------------------------------------------*/
void Fill_Matrix_Db(double* _dev_m, const int _sz, const double _v)
{
	thrust::device_ptr<double> dev_ptr(_dev_m);
	thrust::fill(dev_ptr, dev_ptr + _sz, _v);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*/
/*------------------------------------------------------------------------------------------*/
void Fill_Matrix_Float(float* _dev_m, const int _sz, const float _v)
{
	thrust::device_ptr<float> dev_ptr(_dev_m);
	thrust::fill(dev_ptr, dev_ptr + _sz, _v);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*/
/*------------------------------------------------------------------------------------------*/
void Fill_Matrix_Int(int* _dev_m, const int _sz, const int _v)
{
	thrust::device_ptr<int> dev_ptr(_dev_m);
	thrust::fill(dev_ptr, dev_ptr + _sz, _v);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*/
/*------------------------------------------------------------------------------------------*/