#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include "math.h"
/*------------------------------------------------------------------------------------------*/
#include <stdio.h>
#include <assert.h>
#include <float.h>
#include <limits.h>
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaInit(void);
extern "C" bool cudaDeInit(void);
extern "C" void cudaGetLastError_Sync(void);
/*------------------------------------------------------------------------------------------*/
extern "C" void init_Matrix_Db(double* _dev_m, const int _w, const int _h, const double _d);
extern "C" void init_Matrix_Int(int* _dev_m, const int _w, const int _h, const int _v);
extern "C" void init_Matrix_UL(unsigned long * _dev_m, const int _w, const int _h, const unsigned long _d);
/*------------------------------------------------------------------------------------------*/
#if 0
extern "C" void Fill_Matrix_Int(int* _dev_m, const int _sz, const int _v);
extern "C" void Fill_Matrix_Float(float* _dev_m, const int _sz, const float _v);
extern "C" void Fill_Matrix_Db(double* _dev_m, const int _sz, const double _v);
#endif // 0
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C"  void PerformSuperpixelSLIC_gpu_lab_host_int(
	const int			_kseedsSize,
	const int			_width,
	const int			_height,
	const int*			_host_matrix_l,
	const int*			_host_matrix_a,
	const int*			_host_matrix_b,

	int*				_host_kseedsl,
	int*				_host_kseedsa,
	int*				_host_kseedsb,
	int*				_host_kseedsx,
	int*				_host_kseedsy,
	int*				_host_klabels,
	const unsigned long		_f_m,
	const unsigned long		_f_d);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
__inline__  __device__  unsigned long CalDistLAB_Basic_Square_Int(
	const int	L0,
	const int	A0,
	const int	B0,
	const int	X0,
	const int	Y0,
	const int	L1,
	const int	A1,
	const int	B1,
	const int	X1,
	const int	Y1,
	const unsigned long		_f_m,
	const unsigned long		_f_d)
{
	const unsigned long dst = (L1 - L0)*(L1 - L0) + (B1 - B0)*(B1 - B0) + (A1 - A0)*(A1 - A0);
	const unsigned long dst_xy = (X1 - X0)*(X1 - X0) + (Y1 - Y0)*(Y1 - Y0);
	const unsigned long dist_labxy = dst + dst_xy*_f_m/_f_d;
	return dist_labxy;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void Statistics_Lab_Seed_collect_CPU_Incompletion_host_Int(
	const int			_kseedsSize,
	const int			_width,
	const int			_height,

	const int*		_h_l,
	const int*		_h_a,
	const int*		_h_b,

	const int*			_h_klabels,

	int*				_h_kseed_L,
	int*				_h_kseed_A,
	int*				_h_kseed_B,
	int*				_h_kseed_X,
	int*				_h_kseed_Y,
	int*				_h_kseed_SZ)
{
	const int ZERO = 0;
	const int IMG_SZ = _width*_height;
	const int SEED_BUFF_SZ = _kseedsSize * sizeof(int);
	const int IMG_BUFF_SZ_DB = IMG_SZ * sizeof(double);
	const int IMG_BUFF_SZ_INT = IMG_SZ * sizeof(int);
	const int IMG_BUFF_SZ_UINT = IMG_SZ * sizeof(unsigned int);
	
	memset(_h_kseed_L,				ZERO,	SEED_BUFF_SZ);
	memset(_h_kseed_A,				ZERO,	SEED_BUFF_SZ);
	memset(_h_kseed_B,				ZERO,	SEED_BUFF_SZ);
	memset(_h_kseed_X,				ZERO,	SEED_BUFF_SZ);
	memset(_h_kseed_Y,				ZERO,	SEED_BUFF_SZ);
	memset(_h_kseed_SZ,				ZERO,	SEED_BUFF_SZ);
	
		int ind(0);
		for (register int r = 0; r < _height; r++){
			for (register int c = 0; c < _width; c++){	
				const int label_i = _h_klabels[ind];
						_h_kseed_L[label_i]		+= _h_l[ind];
						_h_kseed_A[label_i]		+= _h_a[ind];
						_h_kseed_B[label_i]		+= _h_b[ind];
						_h_kseed_X[label_i]		+= c;
						_h_kseed_Y[label_i]		+= r;
						_h_kseed_SZ[label_i]	+= 1.0;
						ind++;
			}
		}
			
		for (int k = 0; k < _kseedsSize; k++)
		{
			if (_h_kseed_SZ[k] <= 0) 
				_h_kseed_SZ[k] = 1;

			const double inv_k = 1.0 / _h_kseed_SZ[k];
			//computing inverse now to multiply, than divide later		

			_h_kseed_L[k] *= inv_k;
			_h_kseed_A[k] *= inv_k;
			_h_kseed_B[k] *= inv_k;
			_h_kseed_X[k] *= inv_k;
			_h_kseed_Y[k] *= inv_k;
			
		}
	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void Statistics_Lab_Seed_collect_CPU_Incompletion(
	const int			_kseedsSize,
	const int			_width,
	const int			_height,
	const int*		_h_l,
	const int*		_h_a,
	const int*		_h_b,

	const int*			_dev_klabels,
	int*				_dev_kseed_L,
	int*				_dev_kseed_A,
	int*				_dev_kseed_B,
	int*				_dev_kseed_X,
	int*				_dev_kseed_Y)
{

	const int ZERO = 0;
	const int IMG_SZ = _width*_height;
	const int SEED_BUFF_SZ = _kseedsSize * sizeof(double);
	const int IMG_BUFF_SZ_DB = IMG_SZ * sizeof(double);
	const int IMG_BUFF_SZ_INT = IMG_SZ * sizeof(int);
	const int IMG_BUFF_SZ_UINT = IMG_SZ * sizeof(unsigned int);

	int*				h_klabels =		new int[IMG_SZ];
	int*				h_kseed_L =		new int[_kseedsSize];
	int*				h_kseed_A =		new int[_kseedsSize];
	int*				h_kseed_B =		new int[_kseedsSize];
	int*				h_kseed_X =		new int[_kseedsSize];
	int*				h_kseed_Y =		new int[_kseedsSize];
	int*				h_kseed_SZ =	new int[_kseedsSize];

	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(h_klabels, _dev_klabels, IMG_BUFF_SZ_INT, hipMemcpyDeviceToHost);

	Statistics_Lab_Seed_collect_CPU_Incompletion_host_Int(
		_kseedsSize,
		_width,
		_height,
		_h_l,
		_h_a,
		_h_b,

		h_klabels,
		h_kseed_L,
		h_kseed_A,
		h_kseed_B,
		h_kseed_X,
		h_kseed_Y,
		h_kseed_SZ);

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	cudaStatus = hipMemcpy(_dev_kseed_L,		h_kseed_L, SEED_BUFF_SZ, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(_dev_kseed_A,		h_kseed_A, SEED_BUFF_SZ, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(_dev_kseed_B,		h_kseed_B, SEED_BUFF_SZ, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(_dev_kseed_X,		h_kseed_X, SEED_BUFF_SZ, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(_dev_kseed_Y,		h_kseed_Y, SEED_BUFF_SZ, hipMemcpyHostToDevice);
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	delete[] 		h_klabels;
	delete[] 		h_kseed_L;
	delete[] 		h_kseed_A;
	delete[] 		h_kseed_B;
	delete[] 		h_kseed_X;
	delete[] 		h_kseed_Y;
	delete[] 		h_kseed_SZ;

}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
__inline__ __device__  void SetPixelLablesLab_Int(
	const int* x_offset,
	const int* y_offset,
	const int offsetSize,
	
	const int SeedWH,
	const int labelsX,
	const int labelsY,
	const int x,
	const int y,
	const int m_width,
	const int m_height,
	const int* dev_kseeds_L,
	const int* dev_kseeds_A,
	const int* dev_kseeds_B,
	const int* dev_kseeds_X,
	const int* dev_kseeds_Y,
	const int* dev_L,
	const int* dev_A,
	const int* dev_B,
	int*   klabels,
	unsigned long* distvec,
	const unsigned long		_f_m,
	const unsigned long		_f_d)
{
		unsigned long dist = 0;
		//////////////////////////////////////////////
		const int i = y*m_width + x;
		////////////////////////////////////////////
		for (int xyi = 0; xyi<offsetSize; xyi++) {
			const int nx = labelsX + x_offset[xyi];
			const int ny = labelsY + y_offset[xyi];
			if (nx >= 0 && nx<SeedWH&&ny >= 0 && ny<SeedWH) {
				const int ni = nx + ny*SeedWH;
				{
					 dist = CalDistLAB_Basic_Square_Int(
													dev_L[i], dev_A[i], dev_B[i], x, y,
													dev_kseeds_L[ni], dev_kseeds_A[ni], dev_kseeds_B[ni], dev_kseeds_X[ni], dev_kseeds_Y[ni],
						 							_f_m,
						 							_f_d);

					if (dist < distvec[i])
					{
						distvec[i] = dist;
						klabels[i] = ni;
					}
					
				}
			}

		}

}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnLab_9Seeds_Int(
	const int kseedsSize,
		
	const int _width,
	const int _height,
	
	const int*		dev_L,
	const int*		dev_A,
	const int*		dev_B,

	const int*	dev_kseed_L,
	const int*	dev_kseed_A,
	const int*	dev_kseed_B,
	const int*	dev_kseed_X,
	const int*	dev_kseed_Y,

	int*				klabels,
	unsigned long*		distvec,

	const unsigned long		_f_m,
	const unsigned long		_f_d)
{
	const int x_offset[] = { 0,1,-1,0,0,1,-1,1,-1 };
	const int y_offset[] = { 0,0,0,1,-1,1,-1,-1,1 };

	{
		const int x = blockIdx.x*blockDim.x + threadIdx.x;
		const int y = blockIdx.y*blockDim.y + threadIdx.y;
		const int SeedWH = sqrt((float)kseedsSize);

		const int offsetSize = sizeof(x_offset) / sizeof(int);
		
		if (x >= 0 && x<_width&&y >= 0 && y<_height) {
			const int ind = x + y*_width;
			const int labels_ind = klabels[ind];
			assert(labels_ind >= -1 && labels_ind<kseedsSize);

			int labelsX = 0;
			int labelsY = 0;

			if (labels_ind == -1) {
					const float x_step = 1.0*_width		/	SeedWH;
					const float y_step = 1.0*_height	/	SeedWH;
					labelsX = x / x_step;
					labelsY = y / y_step;				
			}else {
					labelsX = labels_ind	%	SeedWH;
					labelsY = labels_ind	/	SeedWH;
			}
#if 0
			assert(x == dev_X[ind]);
			assert(y == dev_Y[ind]);
#endif // 0



			SetPixelLablesLab_Int(x_offset, y_offset, offsetSize,
					SeedWH, labelsX, labelsY, x, y,
					_width, _height,
					dev_kseed_L, dev_kseed_A, dev_kseed_B, dev_kseed_X, dev_kseed_Y,
					dev_L, dev_A, dev_B,
					klabels,
					distvec,
					_f_m,
					_f_d);

		}

	}

	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void SetMinimumOnLab_9Seeds_Super_Int(
	const int kseedsSize,

	const int _width,
	const int _height,

	const int* dev_L,
	const int* dev_A,
	const int* dev_B,

	int*	dev_kseed_L,
	int*	dev_kseed_A,
	int*	dev_kseed_B,
	int*	dev_kseed_X,
	int*	dev_kseed_Y,

	int*   klabels,
	unsigned long* distvec,

	const unsigned long		_f_m,
	const unsigned long		_f_d)
{
	hipEvent_t start = 0, stop = 0;
	float costtime_ms=0, costtime_us = 0;
	hipError_t cudaStatus_t = hipEventRecord(start, 0);
	{
		dim3    threadsPerBlock(16, 16);
		dim3    numBlock((_width + threadsPerBlock.x - 1) / threadsPerBlock.x, (_height + threadsPerBlock.y - 1) / threadsPerBlock.y);

		SetMinimumOnLab_9Seeds_Int <<<numBlock, threadsPerBlock >>>(
			kseedsSize,
			_width,
			_height,

			dev_L,
			dev_A,
			dev_B,

			dev_kseed_L,
			dev_kseed_A,
			dev_kseed_B,
			dev_kseed_X,
			dev_kseed_Y,

			klabels,
			distvec,

			_f_m,
			_f_d);

		cudaGetLastError_Sync();
	
	}
	cudaStatus_t = hipEventRecord(stop, 0);
	cudaGetLastError_Sync();
	cudaStatus_t = hipEventElapsedTime(&costtime_ms, start, stop);
	costtime_us = costtime_ms * 1000;
	hipEventDestroy(start);
	hipEventDestroy(stop);

}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/

/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_gpu_lab_dev_int(
	const int _kseedsSize,
	const int _width,
	const int _height,
	
	const int*		dev_L,
	const int*		dev_A,
	const int*		dev_B,
	
	int*			_dev_kseedsl,
	int*			_dev_kseedsa,
	int*			_dev_kseedsb,
	int*			_dev_kseedsx,
	int*			_dev_kseedsy,

	int*				_dev_klabels,
	unsigned long*			_dev_distvec,
	const unsigned long		_f_m,
	const unsigned long		_f_d)
{		
		const int SZ = _width*_height;

		init_Matrix_Int(_dev_klabels,_width,_height, -1);
		init_Matrix_UL(_dev_distvec,_width,_height, ULONG_MAX);
		

		SetMinimumOnLab_9Seeds_Super_Int(
			_kseedsSize,
			_width,
			_height,

			dev_L,
			dev_A,
			dev_B,
		
			_dev_kseedsl,
			_dev_kseedsa,
			_dev_kseedsb,
			_dev_kseedsx,
			_dev_kseedsy,

			_dev_klabels,
			_dev_distvec,
			_f_m,
			_f_d);
		
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_gpu_lab_host_int(
	const int			_kseedsSize,
	const int			_width,
	const int			_height,
	const int*			_host_matrix_l,
	const int*			_host_matrix_a,
	const int*			_host_matrix_b,

	int*				_host_kseedsl,
	int*				_host_kseedsa,
	int*				_host_kseedsb,
	int*				_host_kseedsx,
	int*				_host_kseedsy,
	int*				_host_klabels,
	const unsigned long		_f_m,
	const unsigned long		_f_d)
{
	bool cuda_stat_t = cudaInit();
	assert(cuda_stat_t == true);
		{
		const int IMG_SZ = _width*_height;
		const int SEED_BUFF_SZ = _kseedsSize * sizeof(int);
		const int IMG_BUFF_SZ_UL =  IMG_SZ * sizeof(unsigned long);
		const int IMG_BUFF_SZ_INT = IMG_SZ * sizeof(int);
		
					hipError_t cudaStatus;

					int*				dev_kseedsl = nullptr;
					int*				dev_kseedsa = nullptr;
					int*				dev_kseedsb = nullptr;
					int*				dev_kseedsx = nullptr;
					int*				dev_kseedsy = nullptr;

					int*				dev_klabels = nullptr;

					int*				dev_matrix_l = nullptr;
					int*				dev_matrix_a = nullptr;
					int*				dev_matrix_b = nullptr;
					
					unsigned long*		dev_distvec = nullptr;
		
					cudaStatus = hipMalloc((void**)&dev_kseedsl, SEED_BUFF_SZ);
					cudaStatus = hipMalloc((void**)&dev_kseedsa, SEED_BUFF_SZ);
					cudaStatus = hipMalloc((void**)&dev_kseedsb, SEED_BUFF_SZ);
					cudaStatus = hipMalloc((void**)&dev_kseedsx, SEED_BUFF_SZ);
					cudaStatus = hipMalloc((void**)&dev_kseedsy, SEED_BUFF_SZ);

					cudaStatus = hipMalloc((void**)&dev_klabels,  IMG_BUFF_SZ_INT);
					cudaStatus = hipMalloc((void**)&dev_matrix_l, IMG_BUFF_SZ_INT);
					cudaStatus = hipMalloc((void**)&dev_matrix_a, IMG_BUFF_SZ_INT);
					cudaStatus = hipMalloc((void**)&dev_matrix_b, IMG_BUFF_SZ_INT);
					
					cudaStatus = hipMalloc((void**)&dev_distvec, IMG_BUFF_SZ_UL);
						{
							cudaStatus = hipMemcpy(dev_kseedsl,	_host_kseedsl,	SEED_BUFF_SZ,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_kseedsa,	_host_kseedsa,	SEED_BUFF_SZ,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_kseedsb,	_host_kseedsb,	SEED_BUFF_SZ,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_kseedsx,	_host_kseedsx,	SEED_BUFF_SZ,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_kseedsy,	_host_kseedsy,	SEED_BUFF_SZ,	hipMemcpyHostToDevice);
							
							cudaStatus = hipMemcpy(dev_matrix_l,	_host_matrix_l,	IMG_BUFF_SZ_INT,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_matrix_a,	_host_matrix_a, IMG_BUFF_SZ_INT,	hipMemcpyHostToDevice);
							cudaStatus = hipMemcpy(dev_matrix_b,	_host_matrix_b, IMG_BUFF_SZ_INT,	hipMemcpyHostToDevice);
																				
							cudaGetLastError_Sync();
								
							const int ITR_TIMES = 2;

							for (int itr = 0; itr < ITR_TIMES; itr++)
							{
									PerformSuperpixelSLIC_gpu_lab_dev_int(
									_kseedsSize,
									_width,
									_height,

									dev_matrix_l,
									dev_matrix_a,
									dev_matrix_b,

									dev_kseedsl,
									dev_kseedsa,
									dev_kseedsb,
									dev_kseedsx,
									dev_kseedsy,

									dev_klabels,
									dev_distvec,
									_f_m,
									_f_d);

									Statistics_Lab_Seed_collect_CPU_Incompletion(
										_kseedsSize,
										_width,
										_height,

										_host_matrix_l,
										_host_matrix_a,
										_host_matrix_b,

										dev_klabels,
										dev_kseedsl,
										dev_kseedsa,
										dev_kseedsb,
										dev_kseedsx,
										dev_kseedsy);
							}
							

							cudaStatus = hipMemcpy(_host_kseedsl, dev_kseedsl, SEED_BUFF_SZ, hipMemcpyDeviceToHost);
							cudaStatus = hipMemcpy(_host_kseedsa, dev_kseedsa, SEED_BUFF_SZ, hipMemcpyDeviceToHost);
							cudaStatus = hipMemcpy(_host_kseedsb, dev_kseedsb, SEED_BUFF_SZ, hipMemcpyDeviceToHost);
							cudaStatus = hipMemcpy(_host_kseedsx, dev_kseedsx, SEED_BUFF_SZ, hipMemcpyDeviceToHost);
							cudaStatus = hipMemcpy(_host_kseedsy, dev_kseedsy, SEED_BUFF_SZ, hipMemcpyDeviceToHost);
							cudaStatus = hipMemcpy(_host_klabels, dev_klabels, IMG_BUFF_SZ_INT, hipMemcpyDeviceToHost);

						}
					hipFree(dev_kseedsl);
					hipFree(dev_kseedsa);
					hipFree(dev_kseedsb);
					hipFree(dev_kseedsx);
					hipFree(dev_kseedsy);
					hipFree(dev_klabels);
					hipFree(dev_matrix_l);
					hipFree(dev_matrix_a);
					hipFree(dev_matrix_b);
					hipFree(dev_distvec);

		}	
	cuda_stat_t = cudaDeInit();
	assert(cuda_stat_t == true);
}

/*------------------------------------------------------------------------------------------*/
/**
*
*
*/
/*------------------------------------------------------------------------------------------*/