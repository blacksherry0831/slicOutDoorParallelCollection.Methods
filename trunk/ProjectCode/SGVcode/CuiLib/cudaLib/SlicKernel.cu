#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include "math.h"
/*------------------------------------------------------------------------------------------*/
#include <stdio.h>
#include<assert.h>
/*------------------------------------------------------------------------------------------*/
#ifndef UINT32
typedef unsigned int UINT32;
#endif
#if linux||__linux||__linux__ || __GNUC__

#ifndef nullptr
#define nullptr 0
#endif

#ifndef _DEBUG
#define  _DEBUG 1
#endif
#include <unistd.h>    
#include <sys/types.h>  
#include <sys/syscall.h>//Linux system call for thread id
#include <pthread.h>

#endif
/*------------------------------------------------------------------------------------------*/
#ifdef _MSC_VER

#include <windows.h>

#endif
/*------------------------------------------------------------------------------------------*/
extern "C" void Fill_Matrix_Int(int* _dev_m, const int _sz, const int _v);
extern "C" void Fill_Matrix_Float(float* _dev_m, const int _sz, const float _v);
extern "C" void Fill_Matrix_Db(double* _dev_m, const int _sz, const double _v);
/*------------------------------------------------------------------------------------------*/
/**
*时间2014- 11-4
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void PerformSuperpixelSLIC_gpu(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M);
/*------------------------------------------------------------------------------------------*/
extern "C" void PerformSuperpixelSLIC_ThetaMLXY_gpu_simplify(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,   
	const int m_height,
	double*     host_sita_n,
	double*     host_m_n,
	double*     host_L_n,
	double*     host_X_n,
	double*     host_Y_n,
	double*     host_kseedsTheta,
	double*     host_kseedsM,
	double*     host_kseedsL,
	double*     host_kseedsX,
	double*     host_kseedsY,
	int*		host_klabels,
	const int			STEP);
/*------------------------------------------------------------------------------------------*/
/**
*时间2014- 11-4
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void PerformSuperpixelSLIC_gpu_simplify(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M);
extern "C" void PerformSuperpixelSLIC_gpu_simplify2(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void DrawContoursAroundSegments_gpu(
	unsigned int*			ubuff,
	int*					labels,
	const int				width,
	const int				height,
	const unsigned int	color_in,
	const unsigned int  color_out);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void  GetSeedsLabxy_gpu(
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				host_m_lvec,
	double*				host_m_avec,
	double*				host_m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" int GetThreadIdSelfwinlinux(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void Get_Nighbour_E_matrix_gpu(
	int*			labels,
	const  int		NumLabels,
	UINT32 *		Matrix_E,
	const int		width,
	const int		height);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
void InitMaskTemplate(
	unsigned int*			dev_mask,
	int*					dev_labels,
	const int				width,
	const int				height,
	const int                Pixel);
void StatisticsThetaMLXY_Seed_collect_CPU_ALL(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*       host_sita_n,
	double*       host_m_n,
	double*       host_L_n,
	double*       host_X_n,
	double*       host_Y_n,	
	int*				host_klabels,
	unsigned int*       host_img_mask,
    double*	      host_kseed_sita_n,
	double*	      host_kseed_m_n,
	double*	      host_kseed_L_n,
	double*       host_kseed_X_n,
	double*	      host_kseed_Y_n);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaInit_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaDeInit_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void cudaGetLastError_Sync_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
#define  NULL_MATH    1
#define  MATH_USE  0
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/

/*------------------------------------------------------------------------------------------*/
/**
*
*cuda初始化
*@retval true 初始化成功
*@retval false 初始化失败
*/
/*------------------------------------------------------------------------------------------*/
bool cudaInit_CUI(void)
{
		 int i;
		 int device_count;

		 if( hipGetDeviceCount(&device_count) )
		 {
			 printf(" There is zero device beyond 1.0\n"); 
			 return false;
		 }
#if _DEBUG
		//printf("CUDA DEVICE NUMS: %d \n",device_count);
#endif
		 // 找到一个可用的设备
		 for(i=0;i<device_count;i++)
		 {
			  struct hipDeviceProp_t device_prop;
			  if(hipGetDeviceProperties(&device_prop,i)==hipSuccess)
			  {
#if 0
	printf("device properties is :\n"
			"\t device name is %s\n"
			"\t totalGlobalMem is %d\n"
			"\t sharedMemPerBlock is %d\n"
			"\t regsPerBlock is %d\n"
			"\t warpSize is %d\n"
			"\t memPitch is %d\n"
			"\t maxThreadsPerBlock is %d\n"
			"\t maxThreadsDim [3] is %d X %d X %d\n"
			"\t maxGridSize [3] is %d X %d X %d\n"
			"\t totalConstMem is %d\n"
			"\t device version is major %d ,minor %d\n"
			"\t clockRate is %d\n"
			"\t textureAlignment is %d\n"
			"\t deviceOverlap is %d\n"
			"\t multiProcessorCount is %d\n",
			device_prop.name,
			device_prop.totalGlobalMem,
			device_prop.sharedMemPerBlock,
			device_prop.regsPerBlock,
			device_prop.warpSize,
			device_prop.memPitch,
			device_prop.maxThreadsPerBlock,
			device_prop.maxThreadsDim[0],device_prop.maxThreadsDim[1],device_prop.maxThreadsDim[2],
			device_prop.maxGridSize[0],device_prop.maxGridSize[1],device_prop.maxGridSize[2],
			device_prop.totalConstMem,
			device_prop.major,device_prop.minor,
			device_prop.clockRate,
			device_prop.textureAlignment,
			device_prop.deviceOverlap,
			device_prop.multiProcessorCount);
#endif				
					/* if(cudaSetDevice(i)==cudaSuccess){
						 printf("USE GPU ID: %d \n",i);
						 return true;
					 }*/
				break;
			  }

		 }
		 
		 if(i==device_count)
		 {
			  printf("Get the propertites of device occurred error\n");
			  return false;
		 }
		 //有可用设备
		 int default_id,expect_id;		 
		 int thread_id=GetThreadIdSelfwinlinux();
		 expect_id=thread_id%device_count;
		
		 printf("CUDA NUMS: %d, ID: %d,EXPECT: %d \n",device_count,thread_id,expect_id);

		 hipGetDevice(&default_id);

		 if(default_id==expect_id){
			 return true;
		 }else{			 
			 //设备可用
			 if(hipSetDevice(expect_id)==hipSuccess){
				 //printf("USE GPU ID: %d \n",expect_id);
				 return true;
			 }
		 }
		 
		 //////////////////////////////////////////
	//	 int ket=kernelExecTimeoutEnabled();
		 /////////////////////////////////////////
		 return false;
	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*cuda去初始化
*@return 
*@retval true 初始化成功
*@retval false 初始化失败
*/
/*------------------------------------------------------------------------------------------*/
bool cudaDeInit_CUI(void)
{
#if 0
	cudaError_t cudaStatus;
	cudaStatus=cudaDeviceReset();
	if(cudaStatus!=cudaSuccess){
		return false;
	}else{
		return true;
	}
#else
	return true;
#endif
	
}
/*------------------------------------------------------------------------------------------*/
/**
*Cuda等待延时获取错误代码
*
*
*/
/*------------------------------------------------------------------------------------------*/
void cudaGetLastError_Sync_CUI(void)
{
	hipError_t cudaStatus_syn,cudaStatus;
	const char  *errstr=NULL;


	do 
	{
		cudaStatus_syn=hipDeviceSynchronize();
		if (cudaStatus_syn==hipSuccess)
		{
			break;

		}else{
			cudaStatus=hipGetLastError();
			if (cudaStatus!=hipSuccess){
				errstr=hipGetErrorString(cudaStatus);
				printf("cudaDeviceSynchronize: %s",errstr);
			}
		}
		


	} while (cudaStatus_syn!=hipSuccess);

}
/*------------------------------------------------------------------------------------------*/
/**
*
*转换LAB色彩空间成角度、模长、亮度表示法
*
*@param m_width 图像宽度
*@param m_height 图像高度
*@param L 像素点的L分量
*@param A 像素点的A分量
*@param B 像素点的B分量
*@param X 像素点的X坐标
*@param Y 像素点的Y坐标
*@param sita_n 像素点的角度 （归一化到[0,1]）
*@param m_n    像素点的模长 （归一化到[0,1]）
*@param L_n    像素点的亮度 （归一化到[0,1]）
*@param X_n    像素点的X坐标（归一化到[0,1]）
*@param Y_n    像素点的Y坐标（归一化到[0,1]）
*
*/
/*------------------------------------------------------------------------------------------*/
__inline__ __host__ __device__ void ConvertLab2oml(
	int m_width,
	int m_height,
	double L,
	double A,
	double B,
	double X,
	double Y,
	double* sita_n,
	double* m_n,
	double* L_n,
	double* X_n,
	double* Y_n)
{
	assert(L>=0&&L<=100+1);
	assert(A>=-128&&A<=128+1);
	assert(B>=-128&&B<=128+1);
	assert(X>=0&&X<=m_width+1);
	assert(Y>=0&&Y<=m_height+1);
#if 1
	///////////////////////////////////////
	*sita_n=atan2(B,A);
	*sita_n=*sita_n/(2*3.1415927)+0.5;
	///////////////////////////////////////
	*m_n=sqrt(A*A+B*B);
	*m_n=*m_n/(128*1.415);
	///////////////////////////////////////
	*L_n=L/100;
	assert(*L_n>=0&&*L_n<=1+1E-1);
	///////////////////////////////////////
	*X_n=X/m_width;
	assert(*X_n>=0&&*X_n<=1+1E-1);
	*Y_n=Y/m_height;
	assert(*Y_n>=0&&*Y_n<=1+1E-1);
#endif
	assert(*sita_n>=0-0.1&&*sita_n<=1+0.1);
	assert(*m_n>=0-0.1&&*m_n<=1+0.1);
	assert(*L_n>=0-0.1&&*L_n<=1+0.1);
	assert(*X_n>=0-0.1&&*X_n<=1+0.1);
	assert(*Y_n>=0-0.1&&*Y_n<=1+0.1);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*计算两个像素点的距离
*@image html image027.png "色彩公式"
*@image html image029.png "距离公式"
*@image html image031.png "距离综合公式"
*
*
*@param  alpha     角度系数
*@param  betta     模长系数
*@param  gama      亮度系数
*@param  fai       距离系数
*@param  sita_n0   像素0的角度
*@param  m_n0      像素0的模长
*@param  L_n0      像素0的亮度
*@param  X_n0      像素0的等价X坐标
*@param  Y_n0      像素0的等价Y坐标
*@param  sita_n1   像素1的角度
*@param  m_n1      像素1的模长
*@param  L_n1      像素1的亮度
*@param  X_n1      像素1的等价X坐标
*@param  Y_n1      像素1的等价Y坐标
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__inline__ __device__ double  CalculateNewDistance(
	float alpha,
	float betta,
	float gama,
	float fai,
	double sita_n0,
	double m_n0,
	double L_n0,
	double X_n0,
	double Y_n0,
	double sita_n1,
	double m_n1,
	double L_n1,
	double X_n1,
	double Y_n1)
{
	/*alpha=0.6;
	betta=0.3;
	gama=1-alpha-betta;*/
	/*float fai=100;*/
	assert(sita_n0>=0-0.1&&sita_n0<=1+0.1);
	assert(L_n0>=0-0.1&&L_n0<=1+0.1);
	assert(X_n0>=0-0.1&&X_n0<=1+0.1);
	assert(Y_n0>=0-0.1&&Y_n0<=1+0.1);
	assert(sita_n1>=0-0.1&&sita_n1<=1+0.1);
    assert(L_n1>=0-0.1&&L_n1<=1+0.1);
	assert(X_n1>=0-0.1&&X_n1<=1+0.1);
	assert(Y_n1>=0-0.1&&Y_n1<=1+0.1);
#if 0
	double dst=alpha*fabs(sita_n1-sita_n0)+betta*fabs(m_n1-m_n0)+gama*fabs(L_n1-L_n0);
	double dst_xy=fai*sqrt((X_n1-X_n0)*(X_n1-X_n0)+(Y_n1-Y_n0)*(Y_n1-Y_n0));
	dst+=dst_xy;
	return dst;
#else
	double dst=alpha*fabs(sita_n1-sita_n0)+betta*fabs(m_n1-m_n0)+gama*fabs(L_n1-L_n0);
	double dst_xy=fai*(fabs(X_n1-X_n0)+fabs(Y_n1-Y_n0));
	dst+=dst_xy;
	return dst;
#endif
	
}
/*------------------------------------------------------------------------------------------*/
/**
*一个像素与所有种子计算距离保留最小值
*
*@param     kseedsSize 种子个数 
*@param     alpha      角度系数
*@param     betta      模长系数
*@param     gama       亮度系数
*@param     fai        距离参数
*@param     m_width    图像宽度
*@param     m_height   图像高度
*@param   [in]  m_lvec     图像L分量
*@param   [in]  m_avec     图像A分量
*@param   [in] m_bvec     图像B分量
*@param   [in]  dev_sita_n  图像角度分量
*@param   [in] dev_m_n    图像模长分量
*@param   [in]  dev_L_n   图像亮度分量
*@param   [in] dev_X_n   图像X分量
*@param   [in]  dev_Y_n   图像Y分量
*@param   [in]  kseedsl   种子的L分量
*@param   [in] kseedsa   种子的A分量
*@param   [in] kseedsb   种子的B分量
*@param   [in] kseedsx   种子的X坐标分量
*@param   [in] kseedsy   种子的Y坐标分量
*@param   [in] dev_kseed_sita_n  种子角度分量
*@param   [in] dev_kseed_m_n     种子模长分量
*@param   [in] dev_kseed_L_n     种子的L分量
*@param   [in] dev_kseed_X_n     种子的X分量
*@param   [in] dev_kseed_Y_n     种子的Y分量
*@param   [in] klabels           图像的标定信息
*@param   [in] distvec           未使用
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnLab(
	const int kseedsSize,
	float alpha,
	float betta,
	float gama,
	float fai,
	const int m_width,
	const int m_height,
	double* m_lvec,
	double* m_avec,
	double* m_bvec,
	double* kseedsl,
	double* kseedsa,
	double* kseedsb,
	double* kseedsx,
	double* kseedsy,
	int*   klabels,
	double* distvec,
	const int			STEP,
	const double		M)
{

	int x1, y1, x2, y2;
	double l, a, b;
	double dist;
	//double distxy;
	int SpIdx = threadIdx.x+blockIdx.x*blockDim.x;
	int n=SpIdx;
	int offset = STEP;
	if (SpIdx<kseedsSize){
#if 1
///////////////////////////////////////////////
		y1 = max(0.0,			kseedsy[n]-offset);
		y2 = min((double)m_height,	kseedsy[n]+offset);
		x1 = max(0.0,			kseedsx[n]-offset);
		x2 = min((double)m_width,	kseedsx[n]+offset);
	//////////////////////////////////////////////
		for( int y = y1; y < y2; y++ )
			{
				for( int x = x1; x < x2; x++ )
				{
					int i = y*m_width + x;

					l = m_lvec[i];
					a = m_avec[i];
					b = m_bvec[i];
#if 0
					dist =			(l - kseedsl[n])*(l - kseedsl[n]) +
						(a - kseedsa[n])*(a - kseedsa[n]) +
						(b - kseedsb[n])*(b - kseedsb[n]);
					//if (itr>5)
					{
						distxy =		(x - kseedsx[n])*(x - kseedsx[n]) +
							(y - kseedsy[n])*(y - kseedsy[n]);
					}
					//else
					//{
					//	distxy =0;
					//}
					//------------------------------------------------------------------------
					dist += distxy*invwt;//dist = sqrt(dist) + sqrt(distxy*invwt);//this is more exact
					//------------------------------------------------------------------------
#endif
#if 1
					double sita_n, m_n, L_n, X_n, Y_n;
					double Ksita_n, Km_n, KL_n, KX_n, KY_n;
					ConvertLab2oml(m_width,m_height,l,a,b,x,y,&sita_n,&m_n,&L_n,&X_n,&Y_n);
					ConvertLab2oml(m_width,m_height,kseedsl[n],kseedsa[n],kseedsb[n],kseedsx[n],kseedsy[n],&Ksita_n,&Km_n,&KL_n,&KX_n,&KY_n);
					dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
#endif
					if( dist < distvec[i] )
					{
						distvec[i] = dist;
						klabels[i]  = n;
						//atomicExch(&klabels[i],n);
					}
				}
			}

#endif
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*设置超像素图片的Labels
*
*@param   x_offset   围绕中心像素X的偏移量
*@param   y_offset   围绕中心像素Y的偏移量
*@param   offsetSize 偏移数组的长度
*@param   alpha      角度系数
*@param   betta      模长系数
*@param   gama       亮度系数
*@param   fai        距离系数
*@param   SeedWH     种子平方数开根号
*@param   labelsX    超像素块的宽度
*@param   labelsY    超像素块的长度
*@param   x          像素X坐标
*@param   y          像素Y坐标
*@param   m_width    图像宽度
*@param   m_height   图像高度
*@param   dev_kseeds_sita_n  图像的角度矩阵
*@param   dev_kseeds_m_n     图像的宽度矩阵
*@param   dev_kseeds_L_n     图像的亮度矩阵
*@param   dev_kseeds_X_n     图像的等价X矩阵
*@param   dev_kseeds_Y_n     图像的等价Y矩阵
*@param   dev_sita_n         图像的角度分量矩阵
*@param   dev_m_n			 图像的模长分量矩阵
*@param   dev_L_n            图像的亮度分量矩阵
*@param   dev_X_n            图像的X分量矩阵
*@param   dev_Y_n            图像的Y分量矩阵
*@param   klabels            图像的标定信息
*@param   distvec            保存距离最小值的矩阵
*
*/
/*------------------------------------------------------------------------------------------*/
__inline__ __device__  void SetPixelLables(
	const int* x_offset,
	const int* y_offset,
	const int offsetSize,
	const float alpha,
	const float betta,
	const float gama,
	const float fai,
	const int SeedWH,
	const int labelsX,
	const int labelsY,
	const int x,
	const int y,
	const int m_width,
	const int m_height,
	const double* dev_kseeds_sita_n,
	const double* dev_kseeds_m_n,
	const double* dev_kseeds_L_n,
	const double* dev_kseeds_X_n,
	const double* dev_kseeds_Y_n,
	const double* dev_sita_n,
	const double* dev_m_n,
	const double* dev_L_n,
	const double* dev_X_n,
	const double* dev_Y_n,
	int*   klabels,
	double* distvec)
{
#if 0
	{
		double dist;
		//////////////////////////////////////////////
		int i = y*m_width + x;
		////////////////////////////////////////////
		for (int xi=0;xi<offsetSize;xi++){
			for (int yi=0;yi<offsetSize;yi++){
				int nx=labelsX+x_offset[xi];
				int ny=labelsY+y_offset[yi];
				if (nx>=0&&nx<SeedWH&&ny>=0&&ny<SeedWH){
					int ni=nx+ny*SeedWH;
					{
						dist=CalculateNewDistance(alpha,betta,gama,fai,
							dev_sita_n[i],dev_m_n[i],dev_L_n[i],dev_X_n[i],dev_Y_n[i],
							dev_kseeds_sita_n[ni],dev_kseeds_m_n[ni],dev_kseeds_L_n[ni],dev_kseeds_X_n[ni],dev_kseeds_Y_n[ni]);
						if( dist < distvec[i] )
						{
							distvec[i] = dist;
							klabels[i]  = ni;
						}

					}
				}
			}
		}
	}
#endif
#if 1
	{
		double dist;
		//////////////////////////////////////////////
		int i = y*m_width + x;
		////////////////////////////////////////////
		for (int xyi=0;xyi<offsetSize;xyi++){			
				int nx=labelsX+x_offset[xyi];
				int ny=labelsY+y_offset[xyi];
				if (nx>=0&&nx<SeedWH&&ny>=0&&ny<SeedWH){
					int ni=nx+ny*SeedWH;
					{
						dist=CalculateNewDistance(alpha,betta,gama,fai,
							dev_sita_n[i],dev_m_n[i],dev_L_n[i],dev_X_n[i],dev_Y_n[i],
							dev_kseeds_sita_n[ni],dev_kseeds_m_n[ni],dev_kseeds_L_n[ni],dev_kseeds_X_n[ni],dev_kseeds_Y_n[ni]);
						if( dist < distvec[i] )
						{
							distvec[i] = dist;
							klabels[i]  = ni;
						}

					}
				}
			
		}
	}
#endif
}
/*------------------------------------------------------------------------------------------*/
/**
*
*一个像素与5个种子计算距离保留最小值
*
*@param     kseedsSize 种子个数 
*@param     alpha      角度系数
*@param     betta      模长系数
*@param     gama       亮度系数
*@param     fai        距离参数
*@param     m_width    图像宽度
*@param     m_height   图像高度
*@param   [in]  m_lvec     图像L分量
*@param   [in]  m_avec     图像A分量
*@param   [in] m_bvec     图像B分量
*@param   [in]  dev_sita_n  图像角度分量
*@param   [in] dev_m_n    图像模长分量
*@param   [in]  dev_L_n   图像亮度分量
*@param   [in] dev_X_n   图像X分量
*@param   [in]  dev_Y_n   图像Y分量
*@param   [in]  kseedsl   种子的L分量
*@param   [in] kseedsa   种子的A分量
*@param   [in] kseedsb   种子的B分量
*@param   [in] kseedsx   种子的X坐标分量
*@param   [in] kseedsy   种子的Y坐标分量
*@param   [in] dev_kseed_sita_n  种子角度分量
*@param   [in] dev_kseed_m_n     种子模长分量
*@param   [in] dev_kseed_L_n     种子的L分量
*@param   [in] dev_kseed_X_n     种子的X分量
*@param   [in] dev_kseed_Y_n     种子的Y分量
*@param   [in] klabels           图像的标定信息
*@param   [in] distvec           未使用
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnLab_5Seeds(
	const int kseedsSize,
	float alpha,
	float betta,
	float gama,
	float fai,
	const int m_width,
	const int m_height,
	double* m_lvec,
	double* m_avec,
	double* m_bvec,
	double* dev_sita_n,
	double* dev_m_n,
	double* dev_L_n,
	double* dev_X_n,
	double* dev_Y_n,
	double* kseedsl,
	double* kseedsa,
	double* kseedsb,
	double* kseedsx,
	double* kseedsy,
	double*	dev_kseed_sita_n,
	double*	dev_kseed_m_n,
	double*	dev_kseed_L_n,
	double*	dev_kseed_X_n,
	double*	dev_kseed_Y_n,
	int*   klabels,
	double* distvec)
{
	int x_offset[]={0,1,-1,0,0};
	int y_offset[]={0,0,0,1,-1};
////////////////////////////////////////////////////////////////////////
#if 1
{
	int x =blockIdx.x*blockDim.x+threadIdx.x;
	int y =blockIdx.y*blockDim.y+threadIdx.y;
	int SeedWH=sqrt((float)kseedsSize);	
	/////////////////////////////////////////////
	int offsetSize=sizeof(x_offset)/sizeof(int);
	if (x>=0&&x<m_width&&y>=0&&y<m_height){
		int ind=x+y*m_width;
		int labels_ind=klabels[ind];
	assert(labels_ind>=-1&&labels_ind<kseedsSize);
#if 1
		if (labels_ind==-1){
			int x_step=m_width/SeedWH;
			int y_step=m_height/SeedWH;
			int labelsX=x/x_step;
			int labelsY=y/y_step;
			SetPixelLables(x_offset,y_offset,offsetSize,
				alpha,betta,gama,fai,
				SeedWH,labelsX,labelsY,x,y,m_width,m_height,
				dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
				dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,				
				klabels,distvec);
		}else{
			int labelsX=labels_ind%SeedWH;
			int labelsY=labels_ind/SeedWH;
#if 1
			SetPixelLables(x_offset,y_offset,offsetSize,
				alpha,betta,gama,fai,
				SeedWH,labelsX,labelsY,x,y,m_width,m_height,
				dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
				dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,
				klabels,distvec);
#endif	
		}
#endif
	}

}
#endif
////////////////////////////////////////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*一个像素与9个种子计算距离保留最小值
*
*@param     kseedsSize 种子个数 
*@param     alpha      角度系数
*@param     betta      模长系数
*@param     gama       亮度系数
*@param     fai        距离参数
*@param     m_width    图像宽度
*@param     m_height   图像高度
*@param   [in]  m_lvec     图像L分量
*@param   [in]  m_avec     图像A分量
*@param   [in] m_bvec     图像B分量
*@param   [in]  dev_sita_n  图像角度分量
*@param   [in] dev_m_n    图像模长分量
*@param   [in]  dev_L_n   图像亮度分量
*@param   [in] dev_X_n   图像X分量
*@param   [in]  dev_Y_n   图像Y分量
*@param   [in]  kseedsl   种子的L分量
*@param   [in] kseedsa   种子的A分量
*@param   [in] kseedsb   种子的B分量
*@param   [in] kseedsx   种子的X坐标分量
*@param   [in] kseedsy   种子的Y坐标分量
*@param   [in] dev_kseed_sita_n  种子角度分量
*@param   [in] dev_kseed_m_n     种子模长分量
*@param   [in] dev_kseed_L_n     种子的L分量
*@param   [in] dev_kseed_X_n     种子的X分量
*@param   [in] dev_kseed_Y_n     种子的Y分量
*@param   [in] klabels           图像的标定信息
*@param   [in] distvec           未使用
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnLab_9Seeds(
	const int kseedsSize,
	float alpha,
	float betta,
	float gama,
	float fai,
	const int m_width,
	const int m_height,
	double* m_lvec,
	double* m_avec,
	double* m_bvec,
	double* dev_sita_n,
	double* dev_m_n,
	double* dev_L_n,
	double* dev_X_n,
	double* dev_Y_n,
	double* kseedsl,
	double* kseedsa,
	double* kseedsb,
	double* kseedsx,
	double* kseedsy,
	double*	dev_kseed_sita_n,
	double*	dev_kseed_m_n,
	double*	dev_kseed_L_n,
	double*	dev_kseed_X_n,
	double*	dev_kseed_Y_n,
	int*   klabels,
	double* distvec)
{
////////////////////////////////////////////////////////////////////////
	int x_offset[]={0,1,-1,0,0,1,-1,1,-1};
	int y_offset[]={0,0,0,1,-1,1,-1,-1,1};
#if 1
	{
		int x =blockIdx.x*blockDim.x+threadIdx.x;
		int y =blockIdx.y*blockDim.y+threadIdx.y;
		int SeedWH=sqrt((float)kseedsSize);	

		int offsetSize=sizeof(x_offset)/sizeof(int);
		if (x>=0&&x<m_width&&y>=0&&y<m_height){
			int ind=x+y*m_width;
			int labels_ind=klabels[ind];
			assert(labels_ind>=-1&&labels_ind<kseedsSize);
#if 1
			if (labels_ind==-1){
				float x_step=1.0*m_width/SeedWH;
				float y_step=1.0*m_height/SeedWH;
				int labelsX=x/x_step;
				int labelsY=y/y_step;
				SetPixelLables(x_offset,y_offset,offsetSize,
					alpha,betta,gama,fai,
					SeedWH,labelsX,labelsY,x,y,m_width,m_height,
					dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
					dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,				
					klabels,distvec);
			}else{
				int labelsX=labels_ind%SeedWH;
				int labelsY=labels_ind/SeedWH;
#if 1
				SetPixelLables(x_offset,y_offset,offsetSize,
					alpha,betta,gama,fai,
					SeedWH,labelsX,labelsY,x,y,m_width,m_height,
					dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
					dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,
					klabels,distvec);
#endif

			}
#endif
		}

	}
#endif
////////////////////////////////////////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*一个像素与9个种子计算距离保留最小值
*
*@param     kseedsSize 种子个数 
*@param     alpha      角度系数
*@param     betta      模长系数
*@param     gama       亮度系数
*@param     fai        距离参数
*@param     m_width    图像宽度
*@param     m_height   图像高度
*@param   [in]  m_lvec     图像L分量
*@param   [in]  m_avec     图像A分量
*@param   [in] m_bvec     图像B分量
*@param   [in]  dev_sita_n  图像角度分量
*@param   [in] dev_m_n    图像模长分量
*@param   [in]  dev_L_n   图像亮度分量
*@param   [in] dev_X_n   图像X分量
*@param   [in]  dev_Y_n   图像Y分量
*@param   [in]  kseedsl   种子的L分量
*@param   [in] kseedsa   种子的A分量
*@param   [in] kseedsb   种子的B分量
*@param   [in] kseedsx   种子的X坐标分量
*@param   [in] kseedsy   种子的Y坐标分量
*@param   [in] dev_kseed_sita_n  种子角度分量
*@param   [in] dev_kseed_m_n     种子模长分量
*@param   [in] dev_kseed_L_n     种子的L分量
*@param   [in] dev_kseed_X_n     种子的X分量
*@param   [in] dev_kseed_Y_n     种子的Y分量
*@param   [in] klabels           图像的标定信息
*@param   [in] distvec           未使用
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnThetaMLXY_9Seeds(
	const int kseedsSize,
	float alpha,
	float betta,
	float gama,
	float fai,
	const int m_width,
	const int m_height,
	double* dev_sita_n,
	double* dev_m_n,
	double* dev_L_n,
	double* dev_X_n,
	double* dev_Y_n,
	double*	dev_kseed_sita_n,
	double*	dev_kseed_m_n,
	double*	dev_kseed_L_n,
	double*	dev_kseed_X_n,
	double*	dev_kseed_Y_n,
	int*   klabels,
	double* distvec)
{
////////////////////////////////////////////////////////////////////////
	int x_offset[]={0,1,-1,0,0,1,-1,1,-1};
	int y_offset[]={0,0,0,1,-1,1,-1,-1,1};
#if 1
	{
		int x =blockIdx.x*blockDim.x+threadIdx.x;
		int y =blockIdx.y*blockDim.y+threadIdx.y;
		int SeedWH=sqrt((float)kseedsSize);	

		int offsetSize=sizeof(x_offset)/sizeof(int);
		if (x>=0&&x<m_width&&y>=0&&y<m_height){
			int ind=x+y*m_width;
			int labels_ind=klabels[ind];
			assert(labels_ind>=-1&&labels_ind<kseedsSize);
#if 1
			if (labels_ind==-1){
				float x_step=1.0*m_width/SeedWH;
				float y_step=1.0*m_height/SeedWH;
				int labelsX=x/x_step;
				int labelsY=y/y_step;
				SetPixelLables(x_offset,y_offset,offsetSize,
					alpha,betta,gama,fai,
					SeedWH,labelsX,labelsY,x,y,m_width,m_height,
					dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
					dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,				
					klabels,distvec);
			}else{
				int labelsX=labels_ind%SeedWH;
				int labelsY=labels_ind/SeedWH;
#if 1
				SetPixelLables(x_offset,y_offset,offsetSize,
					alpha,betta,gama,fai,
					SeedWH,labelsX,labelsY,x,y,m_width,m_height,
					dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
					dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,
					klabels,distvec);
#endif

			}
#endif
		}

	}
#endif
////////////////////////////////////////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnLab_AllSeeds(
	const int kseedsSize,
	float alpha,
	float betta,
	float gama,
	float fai,
	const int m_width,
	const int m_height,
	double* m_lvec,
	double* m_avec,
	double* m_bvec,
	double* dev_sita_n,
	double* dev_m_n,
	double* dev_L_n,
	double* dev_X_n,
	double* dev_Y_n,
	double* kseedsl,
	double* kseedsa,
	double* kseedsb,
	double* kseedsx,
	double* kseedsy,
	double*	dev_kseed_sita_n,
	double*	dev_kseed_m_n,
	double*	dev_kseed_L_n,
	double*	dev_kseed_X_n,
	double*	dev_kseed_Y_n,
	int*   klabels,
	double* distvec)
{
	////////////////////////////////////////////////////////////////////////
#if 0
	int x =blockIdx.x*blockDim.x+threadIdx.x;
	int y =blockIdx.y*blockDim.y+threadIdx.y;
	int ind=x+y*m_width;
	int labels_ind;
	if (x>=0&&x<m_width&&y>=0&&y<m_height){
		labels_ind= klabels[ind];	
		assert(labels_ind>=-1&&labels_ind<kseedsSize);
		{
			double l, a, b;
			double dist;
			//////////////////////////////////////////////
			int i = y*m_width + x;
			l = m_lvec[i];
			a = m_avec[i];
			b = m_bvec[i];
			double sita_n, m_n, L_n, X_n, Y_n;
			double Ksita_n, Km_n, KL_n, KX_n, KY_n;
			ConvertLab2oml(m_width,m_height,l,a,b,x,y,sita_n,m_n,L_n,X_n,Y_n);
			int kseedareaXmin=max(0,x-STEP);
			int kseedareaYmin=max(0,y-STEP);
			int kseedareaXmax=min(m_width,x+STEP);
			int kseedareaYmax=min(m_height,y+STEP);

			for(int ni=0;ni<kseedsSize;ni++){
				int n=ni;
				int KseedX=kseedsx[n];
				int KseedY=kseedsy[n];
				if ((KseedX>=kseedareaXmin)&&
					(KseedX<kseedareaXmax)&&
					(KseedY>=kseedareaYmin)&&
					(KseedY<kseedareaYmax)){
						ConvertLab2oml(m_width,m_height,kseedsl[n],kseedsa[n],kseedsb[n],kseedsx[n],kseedsy[n],Ksita_n,Km_n,KL_n,KX_n,KY_n);
						dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
						if( dist < distvec[i] )
						{
							distvec[i] = dist;
							klabels[i]  = n;
						}
				}

			}

		}

	}

#endif
	////////////////////////////////////////////////////////////////////////
#if 0
	{
		int x =blockIdx.x*blockDim.x+threadIdx.x;
		int y =blockIdx.y*blockDim.y+threadIdx.y;
		int ind=x+y*m_width;
		int labels_ind;
		if (x>=0&&x<m_width&&y>=0&&y<m_height){
			labels_ind= klabels[ind];	
			assert(labels_ind>=-1&&labels_ind<kseedsSize);
			{
				double l, a, b;
				double dist;
				double sita_n, m_n, L_n, X_n, Y_n;
				double Ksita_n, Km_n, KL_n, KX_n, KY_n;
				//////////////////////////////////////////////
				int i = y*m_width + x;
				////////////////////////////////////////////
				sita_n=dev_sita_n[i];
				m_n=dev_m_n[i];
				L_n=dev_L_n[i];
				X_n=dev_X_n[i];
				Y_n=dev_Y_n[i];
				int kseedareaXmin=max(0,x-STEP);
				int kseedareaYmin=max(0,y-STEP);
				int kseedareaXmax=min(m_width,x+STEP);
				int kseedareaYmax=min(m_height,y+STEP);

				for(int ni=0;ni<kseedsSize;ni++){
					int KseedX=kseedsx[ni];
					int KseedY=kseedsy[ni];
					if ((KseedX>=kseedareaXmin)&&
						(KseedX<kseedareaXmax)&&
						(KseedY>=kseedareaYmin)&&
						(KseedY<kseedareaYmax)){
							Ksita_n=dev_sita_n[ni];
							Km_n=dev_m_n[ni];
							KL_n=dev_L_n[ni];
							KX_n=dev_X_n[ni];
							KY_n=dev_Y_n[ni];
							dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
							if( dist < distvec[i] )
							{
								distvec[i] = dist;
								klabels[i]  = ni;
							}
					}

				}

			}

		}
	}
#endif
	////////////////////////////////////////////////////////////////////////
#if 1
	{
		int x =blockIdx.x*blockDim.x+threadIdx.x;
		int y =blockIdx.y*blockDim.y+threadIdx.y;
		int SeedWH=sqrt((float)kseedsSize);	
		int x_step=m_width/SeedWH;
		int y_step=m_height/SeedWH;
		int STEP=x_step+y_step;
		int ind=x+y*m_width;
		int labels_ind=klabels[ind];
		if (labels_ind==-1){
#if 1
			if (x>=0&&x<m_width&&y>=0&&y<m_height){
				labels_ind= klabels[ind];	
				assert(labels_ind>=-1&&labels_ind<kseedsSize);
				{
					double l, a, b;
					double dist;
					double sita_n, m_n, L_n, X_n, Y_n;
					double Ksita_n, Km_n, KL_n, KX_n, KY_n;
					//////////////////////////////////////////////
					int i = y*m_width + x;
					////////////////////////////////////////////
					sita_n=dev_sita_n[i];
					m_n=dev_m_n[i];
					L_n=dev_L_n[i];
					X_n=dev_X_n[i];
					Y_n=dev_Y_n[i];
					int kseedareaXmin=max(0,x-STEP);
					int kseedareaYmin=max(0,y-STEP);
					int kseedareaXmax=min(m_width,x+STEP);
					int kseedareaYmax=min(m_height,y+STEP);

					for(int ni=0;ni<kseedsSize;ni++){
						int KseedX=kseedsx[ni];
						int KseedY=kseedsy[ni];
						if ((KseedX>=kseedareaXmin)&&
							(KseedX<kseedareaXmax)&&
							(KseedY>=kseedareaYmin)&&
							(KseedY<kseedareaYmax)){
								Ksita_n=dev_sita_n[ni];
								Km_n=dev_m_n[ni];
								KL_n=dev_L_n[ni];
								KX_n=dev_X_n[ni];
								KY_n=dev_Y_n[ni];
								dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
								if( dist < distvec[i] )
								{
									distvec[i] = dist;
									klabels[i]  = ni;
								}
						}
					}
				}
			}
#endif
		}else{
#if 1
			{
				int SeedWH=sqrt((float)kseedsSize);
				int labelsX=labels_ind%SeedWH;
				int labelsY=labels_ind/SeedWH;
				int x_offset[]={0,1,-1,0,0};
				int y_offset[]={0,0,0,1,-1};
				int offsetSize=sizeof(x_offset)/sizeof(int);
				if (x>=0&&x<m_width&&y>=0&&y<m_height){
					assert(labels_ind>=-1&&labels_ind<kseedsSize);
					{
						double l, a, b;
						double dist;
						double sita_n, m_n, L_n, X_n, Y_n;
						double Ksita_n, Km_n, KL_n, KX_n, KY_n;
						//////////////////////////////////////////////
						int i = y*m_width + x;
						////////////////////////////////////////////
						sita_n=dev_sita_n[i];
						m_n=dev_m_n[i];
						L_n=dev_L_n[i];
						X_n=dev_X_n[i];
						Y_n=dev_Y_n[i];
						int kseedareaXmin=max(0,x-STEP);
						int kseedareaYmin=max(0,y-STEP);
						int kseedareaXmax=min(m_width,x+STEP);
						int kseedareaYmax=min(m_height,y+STEP);

						for (int xi=0;xi<offsetSize;xi++){
							for (int yi=0;yi<offsetSize;yi++)
							{
								int nx=labelsX+xi;
								int ny=labelsY+yi;
								if (nx>=0&&nx<SeedWH&&ny>=0&&ny<SeedWH){
									int ni=nx+ny*SeedWH;
									{
										int KseedX=kseedsx[ni];
										int KseedY=kseedsy[ni];
										if ((KseedX>=kseedareaXmin)&&
											(KseedX<kseedareaXmax)&&
											(KseedY>=kseedareaYmin)&&
											(KseedY<kseedareaYmax)){
												Ksita_n=dev_sita_n[ni];
												Km_n=dev_m_n[ni];
												KL_n=dev_L_n[ni];
												KX_n=dev_X_n[ni];
												KY_n=dev_Y_n[ni];
												dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
												if( dist < distvec[i] )
												{
													distvec[i] = dist;
													klabels[i]  = ni;
												}
										}
									}
								}

							}
						}
					}
				}
			}
#endif
		}
	}
#endif
	////////////////////////////////////////////////////////////////////////
#if 0
	{
		int x =blockIdx.x*blockDim.x+threadIdx.x;
		int y =blockIdx.y*blockDim.y+threadIdx.y;
		int SeedWH=sqrt((float)kseedsSize);
		int ind=x+y*m_width;
		int x_offset[]={0,1,-1,0,0};
		int y_offset[]={0,0,0,1,-1};
		int offsetSize=sizeof(x_offset)/sizeof(int);
		int labels_ind=klabels[ind];
		if (labels_ind==-1){
			int x_step=m_width/SeedWH;
			int y_step=m_height/SeedWH;
			int labelsX=x/x_step;
			int labelsY=y/y_step;
#if 1
			if (x>=0&&x<m_width&&y>=0&&y<m_height){
				assert(labels_ind>=-1&&labels_ind<kseedsSize);
				{
					double l, a, b;
					double dist;
					double sita_n, m_n, L_n, X_n, Y_n;
					double Ksita_n, Km_n, KL_n, KX_n, KY_n;
					//////////////////////////////////////////////
					int i = y*m_width + x;
					////////////////////////////////////////////
					sita_n=dev_sita_n[i];
					m_n=dev_m_n[i];
					L_n=dev_L_n[i];
					X_n=dev_X_n[i];
					Y_n=dev_Y_n[i];
					int kseedareaXmin=max(0,x-STEP);
					int kseedareaYmin=max(0,y-STEP);
					int kseedareaXmax=min(m_width,x+STEP);
					int kseedareaYmax=min(m_height,y+STEP);
					for (int xi=0;xi<offsetSize;xi++){
						for (int yi=0;yi<offsetSize;yi++)
						{
							int nx=labelsX+x_offset[xi];
							int ny=labelsY+y_offset[yi];
							if (nx>=0&&nx<SeedWH&&ny>=0&&ny<SeedWH){
								int ni=nx+ny*SeedWH;
								{
									int KseedX=kseedsx[ni];
									int KseedY=kseedsy[ni];
									if ((KseedX>=kseedareaXmin)&&
										(KseedX<kseedareaXmax)&&
										(KseedY>=kseedareaYmin)&&
										(KseedY<kseedareaYmax)){
											Ksita_n=dev_sita_n[ni];
											Km_n=dev_m_n[ni];
											KL_n=dev_L_n[ni];
											KX_n=dev_X_n[ni];
											KY_n=dev_Y_n[ni];
											dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
											if( dist < distvec[i] )
											{
												distvec[i] = dist;
												klabels[i]  = ni;
											}
									}
								}
							}
						}
					}
				}
			}
#endif
		}else{
			int labelsX=labels_ind%SeedWH;
			int labelsY=labels_ind/SeedWH;
#if 1
			if (x>=0&&x<m_width&&y>=0&&y<m_height){
				assert(labels_ind>=-1&&labels_ind<kseedsSize);
				{
					double l, a, b;
					double dist;
					double sita_n, m_n, L_n, X_n, Y_n;
					double Ksita_n, Km_n, KL_n, KX_n, KY_n;
					//////////////////////////////////////////////
					int i = y*m_width + x;
					////////////////////////////////////////////
					sita_n=dev_sita_n[i];
					m_n=dev_m_n[i];
					L_n=dev_L_n[i];
					X_n=dev_X_n[i];
					Y_n=dev_Y_n[i];
					int kseedareaXmin=max(0,x-STEP);
					int kseedareaYmin=max(0,y-STEP);
					int kseedareaXmax=min(m_width,x+STEP);
					int kseedareaYmax=min(m_height,y+STEP);
					for (int xi=0;xi<offsetSize;xi++){
						for (int yi=0;yi<offsetSize;yi++)
						{
							int nx=labelsX+x_offset[xi];
							int ny=labelsY+y_offset[yi];
							if (nx>=0&&nx<SeedWH&&ny>=0&&ny<SeedWH){
								int ni=nx+ny*SeedWH;
								{
									int KseedX=kseedsx[ni];
									int KseedY=kseedsy[ni];
									if ((KseedX>=kseedareaXmin)&&
										(KseedX<kseedareaXmax)&&
										(KseedY>=kseedareaYmin)&&
										(KseedY<kseedareaYmax)){
											Ksita_n=dev_sita_n[ni];
											Km_n=dev_m_n[ni];
											KL_n=dev_L_n[ni];
											KX_n=dev_X_n[ni];
											KY_n=dev_Y_n[ni];
											dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
											if( dist < distvec[i] )
											{
												distvec[i] = dist;
												klabels[i]  = ni;
											}
									}
								}
							}
						}
					}
				}
			}
#endif
		}
	}
#endif
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMatrix(double *matrix,int width,int height,double value_t)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	int ind=i+j*width;
	if(ind<width*height){
		matrix[ind]=value_t;
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMatrix_Efficient(double *matrix,int width,int height,double value_t)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	int offset=blockDim.x*gridDim.x;
	int size_t=width*height;
	/////////////////////////////////////////
	while(ind<size_t){
		matrix[ind]=value_t;
		ind+=offset;
	}
	/////////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600 

#else 
__device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
				__longlong_as_double(assumed)));

		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}
//Read more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#ixzz3HQ3voVZ1 
//Follow us: @GPUComputing on Twitter | NVIDIA on Facebook
#endif
/*------------------------------------------------------------------------------------------*/
/**
*              1*1            
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_blocks(
	const int			kseedSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	int m=blockIdx.x*blockDim.x+threadIdx.x;
	int n=blockIdx.y*blockDim.y+threadIdx.y;
	///////////////////////////////////////
	int m_dim=blockDim.x*gridDim.x;
	int n_dim=blockDim.y*gridDim.y;
	int width_step=width/m_dim;
	int height_step=height/n_dim;
	/////////////////////////////////////////
	int w_start=m*width_step;
	int h_start=n*height_step;
	int w_end=min((m+1)*width_step,width);
	int h_end=min((n+1)*height_step,height);
	///////////////////////////////////////////
	int labels_ind;
	int ind;
	for (int xi=w_start;xi<w_end;xi++){
		for (int yi=h_start;yi<h_end;yi++){
			ind=xi+yi*width;
			labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
#if 1
					{
						atomicAdd(&dev_sigmal[labels_ind],m_lvec[ind]);
						atomicAdd(&dev_sigmaa[labels_ind],m_avec[ind]);
						atomicAdd(&dev_sigmab[labels_ind],m_bvec[ind]);
						atomicAdd(&dev_sigmax[labels_ind],xi);
						atomicAdd(&dev_sigmay[labels_ind],yi);
						atomicAdd(&dev_clustersize[labels_ind],1.0);
					}
#else
						{
							dev_sigmal[labels_ind]+=m_lvec[ind];
							dev_sigmaa[labels_ind]+=m_avec[ind];
							dev_sigmab[labels_ind]+=m_bvec[ind];
							dev_sigmax[labels_ind]+=xi;
							dev_sigmay[labels_ind]+=yi;
							dev_clustersize[labels_ind]+=1.0;
						}
#endif
		}
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*              1*1            
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_blocks_sheard(
	const int			kseedSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	int m=blockIdx.x*blockDim.x+threadIdx.x;
	int n=blockIdx.y*blockDim.y+threadIdx.y;
	///////////////////////////////////////
	int m_dim=blockDim.x*gridDim.x;
	int n_dim=blockDim.y*gridDim.y;
	int width_step=width/m_dim;
	int height_step=height/n_dim;
	/////////////////////////////////////////
	int w_start=m*width_step;
	int h_start=n*height_step;
	int w_end=min((m+1)*width_step,width);
	int h_end=min((n+1)*height_step,height);
	///////////////////////////////////////////
	int labels_ind;
	int ind;
	for (int xi=w_start;xi<w_end;xi++){
		for (int yi=h_start;yi<h_end;yi++){
			ind=xi+yi*width;
			labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
#if 1
			{
				atomicAdd(&dev_sigmal[labels_ind],m_lvec[ind]);
				atomicAdd(&dev_sigmaa[labels_ind],m_avec[ind]);
				atomicAdd(&dev_sigmab[labels_ind],m_bvec[ind]);
				atomicAdd(&dev_sigmax[labels_ind],xi);
				atomicAdd(&dev_sigmay[labels_ind],yi);
				atomicAdd(&dev_clustersize[labels_ind],1.0);
			}
#else
			{
				dev_sigmal[labels_ind]+=m_lvec[ind];
				dev_sigmaa[labels_ind]+=m_avec[ind];
				dev_sigmab[labels_ind]+=m_bvec[ind];
				dev_sigmax[labels_ind]+=xi;
				dev_sigmay[labels_ind]+=yi;
				dev_clustersize[labels_ind]+=1.0;
			}
#endif
		}
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect(
	unsigned int*		mutex,
	const int			kseedSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	int c=blockIdx.x*blockDim.x+threadIdx.x;
	int r=blockIdx.y*blockDim.y+threadIdx.y;
	int ind=c+r*width;
	int labels_ind;
	if (ind>=0&&ind<width*height){
		labels_ind=dev_klabels[ind];
	}else{
		return;
	}

	assert(labels_ind>=0&&labels_ind<kseedSize);
	//	bool finished = false; 
	if (labels_ind>=0&&labels_ind<kseedSize)
	{
		if (ind<width*height){

			//	while(!finished)
			{
				//	if( atomicExch(mutex, 1) == 0)
				{
#if 1
					{
						atomicAdd(&dev_sigmal[labels_ind],m_lvec[ind]);
						atomicAdd(&dev_sigmaa[labels_ind],m_avec[ind]);
						atomicAdd(&dev_sigmab[labels_ind],m_bvec[ind]);
						atomicAdd(&dev_sigmax[labels_ind],c);
						atomicAdd(&dev_sigmay[labels_ind],r);
						atomicAdd(&dev_clustersize[labels_ind],1.0);
					}
#else
					{
						dev_sigmal[labels_ind]+=m_lvec[ind];
						dev_sigmaa[labels_ind]+=m_avec[ind];
						dev_sigmab[labels_ind]+=m_bvec[ind];
						dev_sigmax[labels_ind]+=c;
						dev_sigmay[labels_ind]+=r;
						dev_clustersize[labels_ind]+=1.0;
					}
#endif

					//	finished = true;
					//	atomicExch(mutex, 0);
				}
			}

		}
	}
	/*****************************************/
	//while(!finished)
	//{
	//		if( atomicExch(mutex, 1) == 0){
	//			
	//			
	//			finished = true;
	//			atomicExch(mutex, 0);
	//		}

	//}

	/*****************************************/

}
/*------------------------------------------------------------------------------------------*/
/**
*
*收集全图的种子，默认问K^2（平方数）个数
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_new2014_11_3(
	unsigned int*		mutex,
	const int			kseedSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	__shared__ double temp[4096];
	assert(kseedSize<=4096);
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
			atomicAdd(&temp[labels_ind],m_lvec[ind]);
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_sigmal[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmal[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmal[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmal[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
#if 0
{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
			atomicAdd(&temp[labels_ind],m_avec[ind]);
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_sigmaa[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmaa[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmaa[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmaa[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
				atomicAdd(&temp[labels_ind],m_bvec[ind]);
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_sigmab[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmab[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmab[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmab[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
				atomicAdd(&temp[labels_ind],int(ind%width));
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_sigmax[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmax[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmax[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmax[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
				atomicAdd(&temp[labels_ind],int(ind/width));
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_sigmay[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmay[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmay[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmay[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
				atomicAdd(&temp[labels_ind],1);
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_clustersize[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_clustersize[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_clustersize[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_clustersize[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
#endif	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*收集全图的种子，默认问K^2（平方数）个数
*
*@param			kseedsSize		种子数量
*@param			width		图像宽度
*@param			height		图像高度
*@param	[in]	m_lvec		图像L分量矩阵
*@param	[in]	m_avec		图像A分量矩阵
*@param	[in]	m_bvec		图像B分量矩阵
*@param	[in]	dev_klabels		图像标定矩阵
*@param [in]	dev_img_mask		未使用
*@param	[out]	dev_sigmal		对L分量统计求和
*@param	[out]	dev_sigmaa		对A分量统计求和
*@param	[out]	dev_sigmab		对B分量统计求和
*@param	[out]	dev_sigmax		对X分量统计求和
*@param	[out]	dev_sigmay		对Y分量统计求和
*@param	[out]	dev_clustersize	 未使用 	
*
*/
/*------------------------------------------------------------------------------------------*/
 void StatisticsLABXY_Seed_collect_CPU(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	unsigned int*       dev_img_mask,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	hipError_t cudaStatus;
	unsigned int*       host_img_mask=nullptr;
	int*				host_klabels=new int[width*height];
	double*				host_sigmal=new double[kseedsSize];
	double*				host_sigmaa=new double[kseedsSize];
	double*				host_sigmab=new double[kseedsSize];
	double*				host_sigmax=new double[kseedsSize];
	double*				host_sigmay=new double[kseedsSize];
	double*				host_clustersize=new double[kseedsSize];

#if 1
/////////////////////////////////////////////////////////////////////////////////////////////////////////
	if (dev_img_mask==nullptr){
		host_img_mask=nullptr;
	}else{
		 host_img_mask=new unsigned int[width*height];
		 cudaStatus = hipMemcpy( host_img_mask,dev_img_mask, width*height * sizeof(unsigned int), hipMemcpyDeviceToHost);
	}
	 cudaStatus = hipMemcpy(host_klabels,dev_klabels, width*height * sizeof(int), hipMemcpyDeviceToHost);
	memset(host_sigmal,0,kseedsSize * sizeof(double));
	memset(host_sigmaa,0,kseedsSize * sizeof(double));
	memset(host_sigmab,0,kseedsSize * sizeof(double));
	memset(host_sigmax,0,kseedsSize * sizeof(double));
	memset(host_sigmay,0,kseedsSize * sizeof(double));	
	memset(host_clustersize,0,kseedsSize*sizeof(double));
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif
	{
		int ind(0);
		for(register int r = 0; r < height; r++ )
		{
			for(register  int c = 0; c < width; c++ )
			{
				if (host_img_mask==nullptr){
					host_sigmal[host_klabels[ind]] += m_lvec[ind];
					host_sigmaa[host_klabels[ind]] += m_avec[ind];
					host_sigmab[host_klabels[ind]] += m_bvec[ind];
					host_sigmax[host_klabels[ind]] += c;
					host_sigmay[host_klabels[ind]] += r;
					host_clustersize[host_klabels[ind]] += 1.0;
				}
				if (host_img_mask!=nullptr&&host_img_mask[ind]==0x000000000){
					host_sigmal[host_klabels[ind]] += m_lvec[ind];
					host_sigmaa[host_klabels[ind]] += m_avec[ind];
					host_sigmab[host_klabels[ind]] += m_bvec[ind];
					host_sigmax[host_klabels[ind]] += c;
					host_sigmay[host_klabels[ind]] += r;
					host_clustersize[host_klabels[ind]] += 1.0;
				}				
				//------------------------------------
				//edgesum[klabels[ind]] += edgemag[ind];
				//------------------------------------
				
				ind++;
			}
		}
#if 0
				double x_max=0;
				double y_max=0;
				double *inv=new double[kseedsSize];
				{
					for( int k = 0; k < kseedsSize; k++ )
					{
						if(host_clustersize[k] <= 0 ) host_clustersize[k] = 1;
						inv[k] = 1.0/host_clustersize[k];//computing inverse now to multiply, than divide later
						////////////////////////////////////
						host_sigmal[k] = host_sigmal[k]*inv[k];
						host_sigmaa[k] = host_sigmaa[k]*inv[k];
						host_sigmab[k] = host_sigmab[k]*inv[k];
						host_sigmax[k] = host_sigmax[k]*inv[k];
						host_sigmay[k] = host_sigmay[k]*inv[k];
						if (host_sigmax[k]>x_max){
							x_max=host_sigmax[k];
						}
						if (host_sigmay[k]>y_max){
							y_max=host_sigmay[k];
						}
						//------------------------------------
						//edgesum[k] *= inv[k];
						//------------------------------------
					}
				}
				delete []inv;
#endif
	}
#if 1
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	if (dev_img_mask==nullptr){
		host_img_mask=nullptr;
	}else{
		cudaStatus = hipMemcpy( dev_img_mask,host_img_mask, width*height * sizeof(int), hipMemcpyHostToDevice);
	}
	
	
	cudaStatus = hipMemcpy(dev_sigmal, host_sigmal, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(dev_sigmaa, host_sigmaa, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(dev_sigmab, host_sigmab, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(dev_sigmax, host_sigmax, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(dev_sigmay, host_sigmay, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(dev_clustersize, host_clustersize, kseedsSize * sizeof(double), hipMemcpyHostToDevice);

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif
	delete [] host_img_mask;
	delete [] host_klabels;
	delete [] host_sigmal;
	delete [] host_sigmaa;
	delete [] host_sigmab;
	delete [] host_sigmax;
	delete [] host_sigmay;
	delete [] host_clustersize;
 ;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*收集全图的种子，默认问K^2（平方数）个数
*
*@param			kseedsSize		种子数量
*@param			width		图像宽度
*@param			height		图像高度
*@param	[in]	m_lvec		图像L分量矩阵
*@param	[in]	m_avec		图像A分量矩阵
*@param	[in]	m_bvec		图像B分量矩阵
*@param	[in]	dev_klabels		图像标定矩阵
*@param [in]	dev_img_mask		未使用
*@param	[out]	dev_sigmal		对L分量统计求和
*@param	[out]	dev_sigmaa		对A分量统计求和
*@param	[out]	dev_sigmab		对B分量统计求和
*@param	[out]	dev_sigmax		对X分量统计求和
*@param	[out]	dev_sigmay		对Y分量统计求和
*@param	[out]	dev_clustersize	 未使用 	
*
*/
/*------------------------------------------------------------------------------------------*/
 void StatisticsThetaMLXY_Seed_collect_CPU(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*       host_sita_n,
	double*       host_m_n,
	double*       host_L_n,
	double*       host_X_n,
	double*       host_Y_n,	
	int*				dev_klabels,
    double*	      dev_kseed_sita_n,
	double*	      dev_kseed_m_n,
	double*	      dev_kseed_L_n,
	double*       dev_kseed_X_n,
	double*	      dev_kseed_Y_n)
{
	hipError_t cudaStatus;
	double*	      host_kseed_sita_n=new double[kseedsSize];
	double*	      host_kseed_m_n=new double[kseedsSize];
	double*	      host_kseed_L_n=new double[kseedsSize];
	double*       host_kseed_X_n=new double[kseedsSize];
	double*	      host_kseed_Y_n=new double[kseedsSize];
	int*	      host_klabels=new int[width*height];
	unsigned int*       dev_img_mask;
	unsigned int*       host_img_mask=new unsigned[width*height];
 
   cudaStatus=hipMalloc((void**)&dev_img_mask,sizeof(unsigned int)*width*height);  
   InitMaskTemplate(dev_img_mask,dev_klabels,width,height,4);

   cudaStatus = hipMemcpy(host_kseed_sita_n, dev_kseed_sita_n, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_kseed_m_n, dev_kseed_m_n, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_kseed_L_n, dev_kseed_L_n, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_kseed_X_n, dev_kseed_X_n, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_kseed_Y_n, dev_kseed_Y_n, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_klabels, dev_klabels, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_img_mask,dev_img_mask, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
  
	StatisticsThetaMLXY_Seed_collect_CPU_ALL(
					kseedsSize,
					width,
					height,
					host_sita_n,
					host_m_n,
					host_L_n,
					host_X_n,
					host_Y_n,	
					host_klabels,
					host_img_mask,
					host_kseed_sita_n,
					host_kseed_m_n,
					host_kseed_L_n,
					host_kseed_X_n,
					host_kseed_Y_n);

    delete[]	  host_kseed_sita_n;
	delete[]      host_kseed_m_n;
	delete[]      host_kseed_L_n;
	delete[]      host_kseed_X_n;
	delete[]      host_kseed_Y_n;
	delete[]      host_klabels;
	hipFree(dev_img_mask);
	delete[]      host_img_mask;

}
/*------------------------------------------------------------------------------------------*/
/**
*
*收集全图的种子，默认问K^2（平方数）个数
*
*@param			kseedsSize		种子数量
*@param			width		图像宽度
*@param			height		图像高度
*@param	[in]	m_lvec		图像L分量矩阵
*@param	[in]	m_avec		图像A分量矩阵
*@param	[in]	m_bvec		图像B分量矩阵
*@param	[in]	dev_klabels		图像标定矩阵
*@param [in]	dev_img_mask		未使用
*@param	[out]	dev_sigmal		对L分量统计求和
*@param	[out]	dev_sigmaa		对A分量统计求和
*@param	[out]	dev_sigmab		对B分量统计求和
*@param	[out]	dev_sigmax		对X分量统计求和
*@param	[out]	dev_sigmay		对Y分量统计求和
*@param	[out]	dev_clustersize	 未使用 	
*
*/
/*------------------------------------------------------------------------------------------*/
void StatisticsThetaMLXY_Seed_collect_CPU_ALL(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*       host_sita_n,
	double*       host_m_n,
	double*       host_L_n,
	double*       host_X_n,
	double*       host_Y_n,	
	int*				host_klabels,
	unsigned int*       host_img_mask,
    double*	      host_kseed_sita_n,
	double*	      host_kseed_m_n,
	double*	      host_kseed_L_n,
	double*       host_kseed_X_n,
	double*	      host_kseed_Y_n)
{
      double*			  host_sigmaSita=new double[kseedsSize];
	  double*			  host_sigmaM=new double[kseedsSize];
	  double*			  host_sigmaL=new double[kseedsSize];
	  double*			  host_sigmaX=new double[kseedsSize];
	  double*		      host_sigmaY=new double[kseedsSize];
	  double*			  host_clustersize=new double[kseedsSize];
	memset(host_sigmaSita,0,kseedsSize * sizeof(double));
	memset(host_sigmaM,0,kseedsSize * sizeof(double));
	memset(host_sigmaL,0,kseedsSize * sizeof(double));
	memset(host_sigmaX,0,kseedsSize * sizeof(double));
	memset(host_sigmaY,0,kseedsSize * sizeof(double));	
	memset(host_clustersize,0,kseedsSize*sizeof(double));
	{
		int ind(0);
		for(register int r = 0; r < height; r++ )
		{
			for(register  int c = 0; c < width; c++ )
			{
				if (host_img_mask==nullptr){
					host_sigmaSita[host_klabels[ind]] +=host_sita_n[ind];
					host_sigmaM[host_klabels[ind]] += host_m_n[ind];
					host_sigmaL[host_klabels[ind]] += host_L_n[ind];
					host_sigmaX[host_klabels[ind]] += host_X_n[ind];
					host_sigmaY[host_klabels[ind]] += host_Y_n[ind];
					host_clustersize[host_klabels[ind]] += 1.0;
				}
				if (host_img_mask!=nullptr&&host_img_mask[ind]==0x000000000){
					host_sigmaSita[host_klabels[ind]] +=host_sita_n[ind];
					host_sigmaM[host_klabels[ind]] += host_m_n[ind];
					host_sigmaL[host_klabels[ind]] += host_L_n[ind];
					host_sigmaX[host_klabels[ind]] += host_X_n[ind];
					host_sigmaY[host_klabels[ind]] += host_Y_n[ind];
					host_clustersize[host_klabels[ind]] += 1.0;
				}				
				//------------------------------------
				//edgesum[klabels[ind]] += edgemag[ind];
				//------------------------------------
				
				ind++;
			}
		}
		///////////////////////////////////////////////
		for(int spi=0;spi<kseedsSize;spi++){
			   host_kseed_sita_n[spi]=host_sigmaSita[spi]/host_clustersize[spi];
	           host_kseed_m_n[spi]=host_sigmaM[spi]/host_clustersize[spi];
	           host_kseed_L_n[spi]=host_sigmaL[spi]/host_clustersize[spi];
	           host_kseed_X_n[spi]=host_sigmaX[spi]/host_clustersize[spi];
	           host_kseed_Y_n[spi]=host_sigmaY[spi]/host_clustersize[spi];
		}
	}
	delete []  host_sigmaSita;
	delete []  host_sigmaM;
	delete []  host_sigmaL;
	delete []  host_sigmaX;
	delete []  host_sigmaY;
	delete []  host_clustersize;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*收集全图的种子，默认问K^2（平方数）个数
*
*@param			kseedsSize		种子数量
*@param			width		图像宽度
*@param			height		图像高度
*@param	[in]	m_lvec		图像L分量矩阵
*@param	[in]	m_avec		图像A分量矩阵
*@param	[in]	m_bvec		图像B分量矩阵
*@param	[in]	dev_klabels		图像标定矩阵
*@param [in]	dev_img_mask		未使用
*@param	[out]	dev_sigmal		对L分量统计求和
*@param	[out]	dev_sigmaa		对A分量统计求和
*@param	[out]	dev_sigmab		对B分量统计求和
*@param	[out]	dev_sigmax		对X分量统计求和
*@param	[out]	dev_sigmay		对Y分量统计求和
*@param	[out]	dev_clustersize	 未使用 	
*
*/
/*------------------------------------------------------------------------------------------*/
void StatisticsThetaMLXY_Seed_collect_CPU_Incompletion(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	unsigned int*       dev_img_mask,
	double*	           dev_kseed_sita_n,
	double*	           dev_kseed_m_n,
	double*	           dev_kseed_L_n,
	double*            dev_kseed_X_n,
	double*	           dev_kseed_Y_n)
{
	
	hipError_t cudaStatus;
	unsigned int*       host_img_mask=nullptr;
	int*				host_klabels=new int[width*height];
	double*				host_sigmal=new double[kseedsSize];
	double*				host_sigmaa=new double[kseedsSize];
	double*				host_sigmab=new double[kseedsSize];
	double*				host_sigmax=new double[kseedsSize];
	double*				host_sigmay=new double[kseedsSize];
	double*				host_clustersize=new double[kseedsSize];

#if 1
/////////////////////////////////////////////////////////////////////////////////////////////////////////
	if (dev_img_mask==nullptr){
		host_img_mask=nullptr;
	}else{
		 host_img_mask=new unsigned int[width*height];
		 cudaStatus = hipMemcpy( host_img_mask,dev_img_mask, width*height * sizeof(unsigned int), hipMemcpyDeviceToHost);
	}
	 cudaStatus = hipMemcpy(host_klabels,dev_klabels, width*height * sizeof(int), hipMemcpyDeviceToHost);
	memset(host_sigmal,0,kseedsSize * sizeof(double));
	memset(host_sigmaa,0,kseedsSize * sizeof(double));
	memset(host_sigmab,0,kseedsSize * sizeof(double));
	memset(host_sigmax,0,kseedsSize * sizeof(double));
	memset(host_sigmay,0,kseedsSize * sizeof(double));	
	memset(host_clustersize,0,kseedsSize*sizeof(double));
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif
	{
		int ind(0);
		for(register int r = 0; r < height; r++ )
		{
			for(register  int c = 0; c < width; c++ )
			{
				if (host_img_mask==nullptr){
					host_sigmal[host_klabels[ind]] += m_lvec[ind];
					host_sigmaa[host_klabels[ind]] += m_avec[ind];
					host_sigmab[host_klabels[ind]] += m_bvec[ind];
					host_sigmax[host_klabels[ind]] += c;
					host_sigmay[host_klabels[ind]] += r;
					host_clustersize[host_klabels[ind]] += 1.0;
				}
				if (host_img_mask!=nullptr&&host_img_mask[ind]==0x000000000){
					host_sigmal[host_klabels[ind]] += m_lvec[ind];
					host_sigmaa[host_klabels[ind]] += m_avec[ind];
					host_sigmab[host_klabels[ind]] += m_bvec[ind];
					host_sigmax[host_klabels[ind]] += c;
					host_sigmay[host_klabels[ind]] += r;
					host_clustersize[host_klabels[ind]] += 1.0;
				}				
				//------------------------------------
				//edgesum[klabels[ind]] += edgemag[ind];
				//------------------------------------
				
				ind++;
			}
		}
#if 1
				
				double *inv=new double[kseedsSize];
				{
					for( int k = 0; k < kseedsSize; k++ )
					{
						if(host_clustersize[k] <= 0 ) host_clustersize[k] = 1;
						inv[k] = 1.0/host_clustersize[k];//computing inverse now to multiply, than divide later
						////////////////////////////////////
						host_sigmal[k] = host_sigmal[k]*inv[k];
						host_sigmaa[k] = host_sigmaa[k]*inv[k];
						host_sigmab[k] = host_sigmab[k]*inv[k];
						host_sigmax[k] = host_sigmax[k]*inv[k];
						host_sigmay[k] = host_sigmay[k]*inv[k];
						//------------------------------------
						//edgesum[k] *= inv[k];
						//------------------------------------
							ConvertLab2oml(
							width,
							height,
							host_sigmal[k],
							host_sigmaa[k],
							host_sigmab[k],
							host_sigmax[k],
							host_sigmay[k],
							&host_sigmal[k],
							&host_sigmaa[k],
							&host_sigmab[k],
							&host_sigmax[k],
							&host_sigmay[k]);
					}
				}
				delete []inv;
#endif
	}
#if 1
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	if (dev_img_mask==nullptr){
		host_img_mask=nullptr;
	}else{
		cudaStatus = hipMemcpy( dev_img_mask,host_img_mask, width*height * sizeof(int), hipMemcpyHostToDevice);
	}
	
	cudaStatus = hipMemcpy(dev_kseed_sita_n, host_sigmal, kseedsSize * sizeof(double), hipMemcpyHostToDevice);	
	cudaStatus = hipMemcpy(dev_kseed_m_n, host_sigmaa, kseedsSize * sizeof(double), hipMemcpyHostToDevice);	
	cudaStatus = hipMemcpy( dev_kseed_L_n, host_sigmab, kseedsSize * sizeof(double), hipMemcpyHostToDevice);	
	cudaStatus = hipMemcpy(dev_kseed_X_n, host_sigmax, kseedsSize * sizeof(double), hipMemcpyHostToDevice);	
	cudaStatus = hipMemcpy(dev_kseed_Y_n, host_sigmay, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif
	delete [] host_img_mask;
	delete [] host_klabels;
	delete [] host_sigmal;
	delete [] host_sigmaa;
	delete [] host_sigmab;
	delete [] host_sigmax;
	delete [] host_sigmay;
	delete [] host_clustersize;
 ;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_L(
	const int			kseedSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	__shared__ double temp[4096];
	assert(kseedSize<=4096);
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
		__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
			atomicAdd(&temp[labels_ind],m_lvec[ind]);
			ind+=offset;
		}
		__syncthreads();
		atomicAdd(&dev_sigmal[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmal[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmal[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmal[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*
*收集全图的种子，默认问K^2（平方数）个数
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_gpu_search(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*				kseedsx,
	double*				kseedsy,
	double*				dev_m_lvec,
	double*				dev_m_avec,
	double*				dev_m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	int SeedIdx=threadIdx.x+blockIdx.x*blockDim.x;
	int offset=blockDim.x*gridDim.x;

	while(SeedIdx<kseedsSize){
		////////////////////////////////////////
		int kseedWH=sqrt((float)kseedsSize);
		int x_step=width/(kseedWH);
		int y_step=height/(kseedWH);
		/////////////////////////////////////////
		int x_min=max((int)(kseedsx[SeedIdx]-x_step),(int)0);
		int x_max=min((int)(kseedsx[SeedIdx]+x_step),(int)width);
		int y_min=max((int)(kseedsy[SeedIdx]-y_step),(int)0);
		int y_max=min((int)(kseedsy[SeedIdx]+y_step),(int)height);
		/////////////////////////////////////////

		for (register int xi=x_min;xi<x_max;xi++){
			for (register int yi=y_min;yi<y_max;yi++){
				int ImgIdx=xi+yi*width;
				int labels_t=dev_klabels[ImgIdx];
#if 1
				if (labels_t==SeedIdx){
					dev_sigmal[SeedIdx] += dev_m_lvec[ImgIdx];
					dev_sigmaa[SeedIdx] += dev_m_avec[ImgIdx];
					dev_sigmab[SeedIdx] += dev_m_bvec[ImgIdx];
					dev_sigmax[SeedIdx] += xi;
					dev_sigmay[SeedIdx] += yi;
					dev_clustersize[SeedIdx] += 1.0;
				}
#endif
			}
		}

		
		/////////////////////////////////////////
			SeedIdx+=offset;
	}
	
	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*收集全图的种子，从零开始全图搜索
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_gpu_search_AllImg(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*				dev_m_lvec,
	double*				dev_m_avec,
	double*				dev_m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	int SeedIdx=threadIdx.x+blockIdx.x*blockDim.x;
	int offset=blockDim.x*gridDim.x;
	while(SeedIdx<kseedsSize){
		/*--------------------------------------*/
		int kseedWH=sqrt((float)kseedsSize);
		int x_step=width/(kseedWH);
		int y_step=height/(kseedWH);
		/*--------------------------------------*/
		int x_min=0;
		int x_max=width;
		int y_min=0;
		int y_max=height;
		/*--------------------------------------*/
		for (register int xi=x_min;xi<x_max;xi++){
			for (register int yi=y_min;yi<y_max;yi++){
				int ImgIdx=xi+yi*width;
				int labels_t=dev_klabels[ImgIdx];
#if 1
				if (labels_t==SeedIdx){
					dev_sigmal[SeedIdx] += dev_m_lvec[ImgIdx];
					dev_sigmaa[SeedIdx] += dev_m_avec[ImgIdx];
					dev_sigmab[SeedIdx] += dev_m_bvec[ImgIdx];
					dev_sigmax[SeedIdx] += xi;
					dev_sigmay[SeedIdx] += yi;
					dev_clustersize[SeedIdx] += 1.0;
				}
#endif
			}
		}
		/*--------------------------------------*/
		SeedIdx+=offset;
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*计算新的图块的中心   
*
*
*@param		    	   kseedSize			种子数量	
*@param				    m_width				图像宽度	
*@param				    m_height			图像高度	
*@param		[in]		dev_sigmal			对L分量统计求和
*@param		[in]		dev_sigmaa			图像A分量矩阵
*@param		[in]		dev_sigmab			图像B分量矩阵
*@param		[in]		dev_sigmax			对X分量统计求和
*@param		[in]		dev_sigmay			对Y分量统计求和
*@param		[in]		dev_clustersize		超像素图块的面积大小
*@param		[out]		dev_kseedsl		    种子L分量
*@param		[out]		dev_kseedsa		    图像A分量
*@param		[out]		dev_kseedsb		    图像B分量
*@param		[out]		dev_kseedsx		    种子X分量
*@param		[out]		dev_kseedsy		    种子Y分量
*@param		[out]		dev_kseed_sita_n	图像的角度分量
*@param		[out]		dev_kseed_m_n	    图像的模长分量
*@param		[out]		dev_kseed_L_n	    图像的亮度分量
*@param		[out]		dev_kseed_X_n       图像的X坐标分量
*@param		[out]		dev_kseed_Y_n	    图像的Y坐标分量
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_calculate(
	const int			kseedSize,
	const int			m_width,
	const int			m_height,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize,
	double*				dev_kseedsl,
	double*				dev_kseedsa,
	double*				dev_kseedsb,
	double*				dev_kseedsx,
	double*				dev_kseedsy,
	double*				dev_kseed_sita_n,
	double*				dev_kseed_m_n,
	double*				dev_kseed_L_n,
	double*				dev_kseed_X_n,
	double*				dev_kseed_Y_n)
{
	int ki=blockIdx.x*blockDim.x+threadIdx.x;
	int offset=blockDim.x*gridDim.x;
	while(ki<kseedSize){

		{
				if (dev_clustersize[ki]<=0)  dev_clustersize[ki]=1;
				double inv_k=1.0/dev_clustersize[ki];
				/////////////////////////////////////////////////
				assert(dev_clustersize[ki]>=0&&dev_clustersize[ki]<=m_width*m_height);
				///////////////////////////////////////////////////
				dev_kseedsl[ki] = dev_sigmal[ki]*inv_k;
				dev_kseedsa[ki] = dev_sigmaa[ki]*inv_k;
				dev_kseedsb[ki] = dev_sigmab[ki]*inv_k;
				dev_kseedsx[ki] = dev_sigmax[ki]*inv_k;
				dev_kseedsy[ki] = dev_sigmay[ki]*inv_k;
				///////////////////////////////////////////////////
#if 1
					ConvertLab2oml(
					m_width,
					m_height,
					dev_kseedsl[ki],
					dev_kseedsa[ki],
					dev_kseedsb[ki],
					dev_kseedsx[ki],
					dev_kseedsy[ki],
					&dev_kseed_sita_n[ki],
					&dev_kseed_m_n[ki],
					&dev_kseed_L_n[ki],
					&dev_kseed_X_n[ki],
					&dev_kseed_Y_n[ki]);
#endif
				
		}
		ki+=offset;

	}
	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*@param			kseedsSize    种子个数
*@param			m_width       图像宽度
*@param			m_height      图像高度
*@param			dev_kseedsl   图像的L分量
*@param			dev_kseedsa   图像的A分量
*@param			dev_kseedsb   图像的B分量
*@param			dev_kseedsx   图像的X坐标分量
*@param			dev_kseedsy   图像的X坐标分量
*@param			dev_kseeds_sita_n  图像的角度分量
*@param			dev_kseeds_m_n     图像的模长分量
*@param			dev_kseeds_L_n     图像的亮度分量
*@param			dev_kseeds_X_n     图像的X坐标分量
*@param			dev_kseeds_Y_n     图像的Y坐标分量
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void PerformSuperpixelSLIC_Init_ColorConvert(
	const int			kseedsSize,
	const int			m_width,
	const int			m_height,
	double*				dev_kseedsl,
	double*				dev_kseedsa,
	double*				dev_kseedsb,
	double*				dev_kseedsx,
	double*				dev_kseedsy,
	double*				dev_kseeds_sita_n,
	double*				dev_kseeds_m_n,
	double*				dev_kseeds_L_n,
	double*				dev_kseeds_X_n,
	double*				dev_kseeds_Y_n)
{
	///////////////////////////////////////////
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	int step=blockDim.x*gridDim.x;
	//////////////////////////////////////////
	while(ind<kseedsSize){
			/////////////////////////////////////
#if 1
			ConvertLab2oml(
				m_width,
				m_height,
				dev_kseedsl[ind],
				dev_kseedsa[ind],
				dev_kseedsb[ind],
				dev_kseedsx[ind],
				dev_kseedsy[ind],
				&dev_kseeds_sita_n[ind],
				&dev_kseeds_m_n[ind],
				&dev_kseeds_L_n[ind],
				&dev_kseeds_X_n[ind],
				&dev_kseeds_Y_n[ind]);
#endif
			ind+=step;
	}
	///////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*将LAB色彩装换到角度、模长、亮度 、X、Y组成的分量
*
*@param   kseedsSize  种子个数
*@param   m_width     图像宽度
*@param   m_height    图像高度
*@param  [in]  dev_m_lvec  图像的L分量
*@param  [in] dev_m_avec  图像的A分量
*@param  [in] dev_m_bvec  图像的B分量
*@param  [out] dev_sita_n  图像的角度分量
*@param  [out] dev_m_n	  图像的模长分量
*@param  [out] dev_L_n     图像的亮度分量
*@param  [out] dev_X_n     图像的X坐标分量
*@param  [out] dev_Y_n     图像的Y坐标分量
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void PerformSuperpixelSLIC_ColorConvert(
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				dev_m_lvec,
	double*				dev_m_avec,
	double*				dev_m_bvec,
	double*				dev_sita_n,
	double*				dev_m_n,
	double*				dev_L_n,
	double*				dev_X_n,
	double*				dev_Y_n)
{
	///////////////////////////////////////////
	int xi=blockIdx.x*blockDim.x+threadIdx.x;
	int yj=blockIdx.y*blockDim.y+threadIdx.y;
	int ind=xi+yj*m_width;
	//////////////////////////////////////////
	if(xi<m_width&&
		xi>=0&&
		yj>=0&&
		yj<m_height){
		/////////////////////////////////////
#if 1
			ConvertLab2oml(
				m_width,
				m_height,
				dev_m_lvec[ind],
				dev_m_avec[ind],
				dev_m_bvec[ind],
				xi,
				yj,
				&dev_sita_n[ind],
				&dev_m_n[ind],
				&dev_L_n[ind],
				&dev_X_n[ind],
				&dev_Y_n[ind]);
#endif

		///////////////////////////////////
	}
	///////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*将LAB色彩装换到角度、模长、亮度 、X、Y组成的分量
*
*@param   kseedsSize  种子个数
*@param   m_width     图像宽度
*@param   m_height    图像高度
*@param  [in]  dev_m_lvec  图像的L分量
*@param  [in] dev_m_avec  图像的A分量
*@param  [in] dev_m_bvec  图像的B分量
*@param  [out] dev_sita_n  图像的角度分量
*@param  [out] dev_m_n	  图像的模长分量
*@param  [out] dev_L_n     图像的亮度分量
*@param  [out] dev_X_n     图像的X坐标分量
*@param  [out] dev_Y_n     图像的Y坐标分量
*
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_ColorConvert_gpu(
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				dev_m_lvec,
	double*				dev_m_avec,
	double*				dev_m_bvec,
	double*				dev_sita_n,
	double*				dev_m_n,
	double*				dev_L_n,
	double*				dev_X_n,
	double*				dev_Y_n)
{
#if  _DEBUG
	hipEvent_t start,stop;
	hipError_t cudaStatus;
	float costtime_ms=0;
	float costtime_us=0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	///////////////////////////////////////////////////////////////////////////////////////
	hipEventRecord(start,0);
#endif
	{
		dim3    threadsPerBlock(16,16);
		dim3    numBlock((m_width+threadsPerBlock.x-1)/threadsPerBlock.x,(m_height+threadsPerBlock.y-1)/threadsPerBlock.y);
		PerformSuperpixelSLIC_ColorConvert<<<numBlock,threadsPerBlock>>>(
			kseedsSize,
			m_width,
			m_height,
			dev_m_lvec,
			dev_m_avec,
			dev_m_bvec,
			dev_sita_n,
			dev_m_n,
			dev_L_n,
			dev_X_n,
			dev_Y_n);
	}
	////////////////////////////////////////////////////////////////////////////////////////
	cudaGetLastError_Sync_CUI();
#if _DEBUG
	cudaStatus=hipEventRecord(stop,0);	
	cudaGetLastError_Sync_CUI();
	cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
	costtime_us=costtime_ms*1000;
#endif
	cudaGetLastError_Sync_CUI();
#if _DEBUG
	hipEventDestroy(start);
	hipEventDestroy(stop);
#endif

}
/*------------------------------------------------------------------------------------------*/
/**
*利用GPU实现超像素算法
*
*@param			alpha			角度系数
*@param			betta			模长系数
*@param			gama			亮度系数
*@param			fai				距离参数
*@param			kseedsSize		种子个数 
*@param			m_width			图像宽度
*@param			m_height		图像高度
*@param		[]	host_m_lvec		图像L分量
*@param		[]	host_m_avec		图像A分量
*@param		[]	host_m_bvec		图像B分量
*@param		[]	host_kseedsl	种子的L分量
*@param		[]	host_kseedsa	种子的A分量
*@param		[]	host_kseedsb	种子的B分量
*@param		[]	host_kseedsx	种子的X坐标分量
*@param		[]	host_kseedsy	种子的Y坐标分量
*@param		[]	host_klabels	图像的标定信息
*@param			STEP
*@param		[]	host_edgemag
*@param  		M
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_gpu(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				host_m_lvec,
	double*				host_m_avec,
	double*				host_m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M)
{
	hipError_t cudaStatus;
	///////////////////////////////////////////
	double*				dev_kseedsl;
	double*				dev_kseedsa;
	double*				dev_kseedsb;
	double*				dev_kseedsx;
	double*				dev_kseedsy;
	int*				dev_klabels;
	double*				dev_m_lvec;
	double*				dev_m_avec;
	double*				dev_m_bvec;
	double*				dev_sita_n;
	double*				dev_m_n;
	double*				dev_L_n;
	double*				dev_X_n;
	double*				dev_Y_n;
	double*				dev_edgemag=nullptr;
	///////////////////////////////////////////
	double* dev_distvec;
	///////////////////////////////////////////
	assert(cudaInit_CUI()==true);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	int blocks=prop.multiProcessorCount;

	{
		cudaStatus = hipMalloc((void**)&dev_kseedsl, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsb, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsx, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsy, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_klabels,m_width*m_height * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_m_lvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_avec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_bvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_distvec, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_sita_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_L_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_X_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_Y_n, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMemcpy(dev_kseedsl, host_kseedsl, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsa, host_kseedsa, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsb, host_kseedsb, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsx, host_kseedsx, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsy, host_kseedsy, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_klabels, host_klabels, m_width*m_height * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_lvec, host_m_lvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_avec, host_m_avec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_bvec, host_m_bvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		////////////////////////////////////////////////////////////////////////////////
	}
	double* dev_sigmal;
	double* dev_sigmaa;
	double* dev_sigmab;
	double* dev_sigmax;
	double* dev_sigmay;
	double* dev_clustersize;
	/////////////////////////////////////////////////////////////////////////////////////
	double*  dev_kseed_sita_n;
	double*  dev_kseed_m_n;
	double*  dev_kseed_L_n;
	double*  dev_kseed_X_n;
	double*  dev_kseed_Y_n;
	/////////////////////////////////////////////////////////////////////////////////////
	{
		cudaStatus = hipMalloc((void**)&dev_sigmal, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmaa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmab, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmax, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmay, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_clustersize, kseedsSize * sizeof(double));
		cudaStatus = hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
		cudaGetLastError_Sync_CUI();
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize * sizeof(double));
	}
	hipEvent_t start,stop;
	float costtime_ms=0;
	float costtime_us=0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#if 1
	///////////////////////////////////////////
	PerformSuperpixelSLIC_ColorConvert_gpu(
		kseedsSize,
		m_width,
		m_height,
		dev_m_lvec,
		dev_m_avec,
		dev_m_bvec,
		dev_sita_n,
		dev_m_n,
		dev_L_n,
		dev_X_n,
		dev_Y_n);
	//////////////////////////////////////////
	PerformSuperpixelSLIC_Init_ColorConvert<<<1,256>>>(
		kseedsSize,
		m_width,
		m_height,
		dev_kseedsl,
		dev_kseedsa,
		dev_kseedsb,
		dev_kseedsx,
		dev_kseedsy,
		dev_kseed_sita_n,
		dev_kseed_m_n,
		dev_kseed_L_n,
		dev_kseed_X_n,
		dev_kseed_Y_n);
	//////////////////////////////////////////
#endif

	for( int itr = 0; itr < 10; itr++ )
	{
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		
		{
			cudaStatus=hipEventRecord(start,0);
#if 0
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+15)/threadsPerBlock.x,(m_height+15)/threadsPerBlock.y);
			SetMatrix<<<numBlock,threadsPerBlock>>>(dev_distvec,m_width,m_height,1.7976931348623158e+308);
#else
			SetMatrix_Efficient<<<16,16*16>>>(dev_distvec,m_width,m_height,1.7976931348623158e+308);
#endif
			
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}
		
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{
			cudaStatus=hipEventRecord(start,0);
#if 0
			SetMinimumOnLab<<<(kseedsSize+511)/512,512>>>(
				kseedsSize,
				alpha,
				betta,
				gama,
				fai,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_kseedsl,
				dev_kseedsa,
				dev_kseedsb,
				dev_kseedsx,
				dev_kseedsy,
				dev_klabels,
				dev_distvec,
				STEP,
				M);
#else
			dim3    threadsPerBlock(32,32);
			dim3    numBlock((m_width+31)/threadsPerBlock.x,(m_height+31)/threadsPerBlock.y);
			SetMinimumOnLab_AllSeeds<<<numBlock,threadsPerBlock>>>(
				kseedsSize,
				alpha,
				betta,
				gama,
				fai,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_sita_n,
				dev_m_n,
				dev_L_n,
				dev_X_n,
				dev_Y_n,
				dev_kseedsl,
				dev_kseedsa,
				dev_kseedsb,
				dev_kseedsx,
				dev_kseedsy,
				dev_kseed_sita_n,
				dev_kseed_m_n,
				dev_kseed_L_n,
				dev_kseed_X_n,
				dev_kseed_Y_n,
				dev_klabels,
				dev_distvec);
#endif
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
		}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{
			/*cudaStatus=cudaEventRecord(start,0);*/
			/*---------------------------------------------------------------------------------*/
//			const int numk=kseedsSize;
			hipMemset(dev_sigmal,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmaa,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmab,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmax,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmay,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			/*----------------------------------------------------------------------------------*/
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+15)/threadsPerBlock.x,(m_height+15)/threadsPerBlock.y);
			unsigned int *mutex; 
			unsigned int zero = 0; 
			hipMalloc( (void**)&mutex, sizeof(unsigned int) ); 
			hipMemcpy(mutex, &zero, sizeof(unsigned int), hipMemcpyHostToDevice );
#if 0
			cudaStatus=cudaEventRecord(start,0);
			//1225.2045ms
			StatisticsLABXY_Seed_collect<<<numBlock,threadsPerBlock>>>(
				mutex,
				kseedsSize,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
			cudaGetLastError_Sync_CUI();
			cudaStatus=cudaEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=cudaEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
#endif
#if 0
			//69.114082ms
			StatisticsLABXY_Seed_collect_CPU(
				kseedsSize,
				m_width,
				m_height,
				host_m_lvec,
				host_m_avec,
				host_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
#endif
#if 1
			cudaStatus=hipEventRecord(start,0);
			//344.65720ms
			StatisticsLABXY_Seed_collect_new2014_11_3<<<blocks*2,1024>>>(
				mutex,
				kseedsSize,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
#endif
#if 0
			cudaStatus=cudaEventRecord(start,0);
			dim3    threadsPerBlock_Quad(32,32);
			dim3    numBlock_Quad(4,4);
			StatisticsLABXY_Seed_collect_blocks<<<numBlock_Quad,threadsPerBlock_Quad>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
			cudaGetLastError_Sync_CUI();
			cudaStatus=cudaEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=cudaEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;

#endif
			cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
			StatisticsLABXY_Seed_calculate<<<1,512>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize,
				dev_kseedsl,
				dev_kseedsa,
				dev_kseedsb,
				dev_kseedsx,
				dev_kseedsy,
				dev_kseed_sita_n,
				dev_kseed_m_n,
				dev_kseed_L_n,
				dev_kseed_X_n,
				dev_kseed_Y_n);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
		}
		
	}
	hipMemcpy(host_klabels,dev_klabels,m_width*m_height*sizeof(int),hipMemcpyDeviceToHost);
	{
	hipFree(dev_kseedsl);
	hipFree(dev_kseedsa);
	hipFree(dev_kseedsb);
	hipFree(dev_kseedsx);
	hipFree(dev_kseedsy);
	hipFree(dev_klabels);
	hipFree(dev_m_lvec);
	hipFree(dev_m_avec);
	hipFree(dev_m_bvec);
	hipFree(dev_distvec);
	////////////////////////
	hipFree(dev_sita_n);
	hipFree(dev_m_n);
	hipFree(dev_L_n);
	hipFree(dev_X_n);
	hipFree(dev_Y_n);
		{
			hipFree(dev_sigmal);
			hipFree(dev_sigmaa);
			hipFree(dev_sigmab);
			hipFree(dev_sigmax);
			hipFree(dev_sigmay);
			hipFree(dev_clustersize);
			//////////////////////////
			hipFree(dev_kseed_sita_n) ;
			hipFree(dev_kseed_m_n);
			hipFree(dev_kseed_L_n);
			hipFree(dev_kseed_X_n);
			hipFree(dev_kseed_Y_n);
		}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	assert(cudaDeInit_CUI()==true);
	
}
/*------------------------------------------------------------------------------------------*/
/**
*cuda方法实行超像素算法。
*
*
*@param    alpha  角度参数alpha
*@param    betta  模值betta
*@param    gama   亮度gama
*@param    fai    XY距离
*@param    kseedsSize  种子个数
*@param    m_width   图像宽度
*@param    m_height  图像高度
*@param	   m_lvec  图像L分量
*@param	   m_avec  图像A分量
*@param	   m_bvec  图像B分量
*@param	   kseedsl  种子L分量
*@param	   kseedsa  种子A分量
*@param	   kseedsb  种子B分量
*@param	   kseedsx  种子X分量
*@param	   kseedsy  种子Y分量
*@param	   klabels  图像图块的标定图像
*@param	   STEP     未使用
*@param	   edgemag  未使用
*@param	   M        未使用密度因素
*
*
*@note
*利用GPU实现超像素算法
*小图1.8S  大图7.2S
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_gpu_simplify(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,   
	const int m_height,
	double*				host_m_lvec,
	double*				host_m_avec,
	double*				host_m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M)
{
	hipError_t cudaStatus;
	///////////////////////////////////////////
	double*				dev_kseedsl;
	double*				dev_kseedsa;
	double*				dev_kseedsb;
	double*				dev_kseedsx;
	double*				dev_kseedsy;
	int*				dev_klabels;
	double*				dev_m_lvec;
	double*				dev_m_avec;
	double*				dev_m_bvec;
	double*				dev_sita_n;
	double*				dev_m_n;
	double*				dev_L_n;
	double*				dev_X_n;
	double*				dev_Y_n;
	double*				dev_edgemag=nullptr;
	///////////////////////////////////////////
	double* dev_distvec;
	///////////////////////////////////////////
	assert(cudaInit_CUI()==true);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	int blocks=prop.multiProcessorCount;
	{
		cudaStatus = hipMalloc((void**)&dev_kseedsl, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsb, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsx, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsy, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_klabels,m_width*m_height * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_m_lvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_avec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_bvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_distvec, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_sita_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_L_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_X_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_Y_n, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMemcpy(dev_kseedsl, host_kseedsl, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsa, host_kseedsa, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsb, host_kseedsb, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsx, host_kseedsx, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsy, host_kseedsy, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_klabels, host_klabels, m_width*m_height * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_lvec, host_m_lvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_avec, host_m_avec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_bvec, host_m_bvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		////////////////////////////////////////////////////////////////////////////////
	}
	double* dev_sigmal;
	double* dev_sigmaa;
	double* dev_sigmab;
	double* dev_sigmax;
	double* dev_sigmay;
	double* dev_clustersize;
	/////////////////////////////////////////////////////////////////////////////////////
	double*  dev_kseed_sita_n;
	double*  dev_kseed_m_n;
	double*  dev_kseed_L_n;
	double*  dev_kseed_X_n;
	double*  dev_kseed_Y_n;
	/////////////////////////////////////////////////////////////////////////////////////
	{
		cudaStatus = hipMalloc((void**)&dev_sigmal, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmaa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmab, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmax, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmay, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_clustersize, kseedsSize * sizeof(double));
		cudaStatus=hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
		cudaGetLastError_Sync_CUI();
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize * sizeof(double));
	}
	hipEvent_t start,stop;
	float costtime_ms=0;
	float costtime_us=0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#if 1
	///////////////////////////////////////////
	PerformSuperpixelSLIC_ColorConvert_gpu(
		kseedsSize,
		m_width,
		m_height,
		dev_m_lvec,
		dev_m_avec,
		dev_m_bvec,
		dev_sita_n,
		dev_m_n,
		dev_L_n,
		dev_X_n,
		dev_Y_n);
	//////////////////////////////////////////
	PerformSuperpixelSLIC_Init_ColorConvert<<<1,256>>>(
		kseedsSize,
		m_width,
		m_height,
		dev_kseedsl,
		dev_kseedsa,
		dev_kseedsb,
		dev_kseedsx,
		dev_kseedsy,
		dev_kseed_sita_n,
		dev_kseed_m_n,
		dev_kseed_L_n,
		dev_kseed_X_n,
		dev_kseed_Y_n);
	cudaGetLastError_Sync_CUI();
	//////////////////////////////////////////
#endif
	for( int itr = 0; itr <10; itr++ )
	{
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		{
			cudaStatus=hipEventRecord(start,0);
			SetMatrix_Efficient<<<16,16*16>>>(dev_distvec,m_width,m_height,1.7976931348623158e+308);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}
		
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{
			cudaStatus=hipEventRecord(start,0);
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+threadsPerBlock.x-1)/threadsPerBlock.x,(m_height+threadsPerBlock.y-1)/threadsPerBlock.y);

			SetMinimumOnThetaMLXY_9Seeds<<<numBlock,threadsPerBlock>>>(		
					kseedsSize,
					alpha,
					betta,
					gama,
					fai,
					m_width,
					m_height,			
					dev_sita_n,
					dev_m_n,
					dev_L_n,
					dev_X_n,
					dev_Y_n,				
					dev_kseed_sita_n,
					dev_kseed_m_n,
					dev_kseed_L_n,
					dev_kseed_X_n,
					dev_kseed_Y_n,
					dev_klabels,
					dev_distvec);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}

		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{			
#if 1
			cudaStatus=hipEventRecord(start,0);
			/*---------------------------------------------------------------------------------*/
			//			const int numk=kseedsSize;
			hipMemset(dev_sigmal,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmaa,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmab,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmax,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmay,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			/*----------------------------------------------------------------------------------*/
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+15)/threadsPerBlock.x,(m_height+15)/threadsPerBlock.y);
			cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
//69.114082ms
			StatisticsLABXY_Seed_collect_CPU(
				kseedsSize,
				m_width,
				m_height,
				host_m_lvec,
				host_m_avec,
				host_m_bvec,
				dev_klabels,
				nullptr,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
						cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
			StatisticsLABXY_Seed_calculate<<<1,512>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize,
				dev_kseedsl,
				dev_kseedsa,
				dev_kseedsb,
				dev_kseedsx,
				dev_kseedsy,
				dev_kseed_sita_n,
				dev_kseed_m_n,
				dev_kseed_L_n,
				dev_kseed_X_n,
				dev_kseed_Y_n);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
			
#else

			cudaStatus=cudaEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
			StatisticsLABXY_Seed_collect_gpu_search<<<(kseedsSize+511)/512,512>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_kseedsx,
				dev_kseedsy,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
			cudaGetLastError_Sync_CUI();
			cudaStatus=cudaEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=cudaEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
#endif
		}
	}
#if 1
	////////////////////////////////////////////////////////////////////////////////////////
	cudaStatus=hipMemcpy(host_klabels,dev_klabels,m_width*m_height*sizeof(int),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsl,dev_kseedsl, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsa,dev_kseedsa, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsb,dev_kseedsb, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsx,dev_kseedsx, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsy,dev_kseedsy, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);
#endif
	///////////////////////////////////////////////////////////////////////////////////////
	{
		hipFree(dev_kseedsl);
		hipFree(dev_kseedsa);
		hipFree(dev_kseedsb);
		hipFree(dev_kseedsx);
		hipFree(dev_kseedsy);
		hipFree(dev_klabels);
		hipFree(dev_m_lvec);
		hipFree(dev_m_avec);
		hipFree(dev_m_bvec);
		hipFree(dev_distvec);
		////////////////////////
		hipFree(dev_sita_n);
		hipFree(dev_m_n);
		hipFree(dev_L_n);
		hipFree(dev_X_n);
		hipFree(dev_Y_n);
		{
			hipFree(dev_sigmal);
			hipFree(dev_sigmaa);
			hipFree(dev_sigmab);
			hipFree(dev_sigmax);
			hipFree(dev_sigmay);
			hipFree(dev_clustersize);
			//////////////////////////
			hipFree(dev_kseed_sita_n) ;
			hipFree(dev_kseed_m_n);
			hipFree(dev_kseed_L_n);
			hipFree(dev_kseed_X_n);
			hipFree(dev_kseed_Y_n);
		}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	assert(cudaDeInit_CUI()==true);
}
/*------------------------------------------------------------------------------------------*/
/**
*cuda方法实行超像素算法。
*
*
*@param    alpha  角度参数alpha
*@param    betta  模值betta
*@param    gama   亮度gama
*@param    fai    XY距离
*@param    kseedsSize  种子个数
*@param    m_width   图像宽度
*@param    m_height  图像高度
*@param	   m_lvec  图像L分量
*@param	   m_avec  图像A分量
*@param	   m_bvec  图像B分量
*@param	   kseedsl  种子L分量
*@param	   kseedsa  种子A分量
*@param	   kseedsb  种子B分量
*@param	   kseedsx  种子X分量
*@param	   kseedsy  种子Y分量
*@param	   klabels  图像图块的标定图像
*@param	   STEP     未使用
*@param	   edgemag  未使用
*@param	   M        未使用密度因素
*
*
*@note
*利用GPU实现超像素算法
*小图1.8S  大图7.2S
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_gpu_simplify2(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,   
	const int m_height,
	double*				host_m_lvec,
	double*				host_m_avec,
	double*				host_m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M)
{
	hipError_t cudaStatus;
	///////////////////////////////////////////
	double*				dev_kseedsl;
	double*				dev_kseedsa;
	double*				dev_kseedsb;
	double*				dev_kseedsx;
	double*				dev_kseedsy;
	int*				dev_klabels;
	double*				dev_m_lvec;
	double*				dev_m_avec;
	double*				dev_m_bvec;
	double*				dev_sita_n;
	double*				dev_m_n;
	double*				dev_L_n;
	double*				dev_X_n;
	double*				dev_Y_n;
	double*				dev_edgemag=nullptr;
	///////////////////////////////////////////
	double* dev_distvec;
	///////////////////////////////////////////
	assert(cudaInit_CUI()==true);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	int blocks=prop.multiProcessorCount;

	{
		cudaStatus = hipMalloc((void**)&dev_kseedsl, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsb, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsx, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsy, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_klabels,m_width*m_height * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_m_lvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_avec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_bvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_distvec, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_sita_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_L_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_X_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_Y_n, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMemcpy(dev_kseedsl, host_kseedsl, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsa, host_kseedsa, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsb, host_kseedsb, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsx, host_kseedsx, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsy, host_kseedsy, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_klabels, host_klabels, m_width*m_height * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_lvec, host_m_lvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_avec, host_m_avec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_bvec, host_m_bvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		////////////////////////////////////////////////////////////////////////////////
	}

	double* dev_clustersize;
	/////////////////////////////////////////////////////////////////////////////////////
	double*  dev_kseed_sita_n;
	double*  dev_kseed_m_n;
	double*  dev_kseed_L_n;
	double*  dev_kseed_X_n;
	double*  dev_kseed_Y_n;
	/////////////////////////////////////////////////////////////////////////////////////
	{

		cudaStatus = hipMalloc((void**)&dev_clustersize, kseedsSize * sizeof(double));
		cudaStatus=hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
		cudaGetLastError_Sync_CUI();
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize * sizeof(double));
	}
	hipEvent_t start,stop;
	float costtime_ms=0;
	float costtime_us=0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#if 1
	///////////////////////////////////////////
	PerformSuperpixelSLIC_ColorConvert_gpu(
		kseedsSize,
		m_width,
		m_height,
		dev_m_lvec,
		dev_m_avec,
		dev_m_bvec,
		dev_sita_n,
		dev_m_n,
		dev_L_n,
		dev_X_n,
		dev_Y_n);
	//////////////////////////////////////////
	PerformSuperpixelSLIC_Init_ColorConvert<<<1,256>>>(
		kseedsSize,
		m_width,
		m_height,
		dev_kseedsl,
		dev_kseedsa,
		dev_kseedsb,
		dev_kseedsx,
		dev_kseedsy,
		dev_kseed_sita_n,
		dev_kseed_m_n,
		dev_kseed_L_n,
		dev_kseed_X_n,
		dev_kseed_Y_n);
	cudaGetLastError_Sync_CUI();
	//////////////////////////////////////////
#endif
	for( int itr = 0; itr <2; itr++ )
	{
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		{
			cudaStatus=hipEventRecord(start,0);
			SetMatrix_Efficient<<<16,16*16>>>(dev_distvec,m_width,m_height,1.7976931348623158e+308);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}
		
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{
			cudaStatus=hipEventRecord(start,0);
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+threadsPerBlock.x-1)/threadsPerBlock.x,(m_height+threadsPerBlock.y-1)/threadsPerBlock.y);

			SetMinimumOnThetaMLXY_9Seeds<<<numBlock,threadsPerBlock>>>(		
					kseedsSize,
					alpha,
					betta,
					gama,
					fai,
					m_width,
					m_height,			
					dev_sita_n,
					dev_m_n,
					dev_L_n,
					dev_X_n,
					dev_Y_n,				
					dev_kseed_sita_n,
					dev_kseed_m_n,
					dev_kseed_L_n,
					dev_kseed_X_n,
					dev_kseed_Y_n,
					dev_klabels,
					dev_distvec);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}

		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{			

			StatisticsThetaMLXY_Seed_collect_CPU_Incompletion(
								kseedsSize,
									m_width,
									m_height,
									host_m_lvec,
									host_m_avec,
									host_m_bvec,
									dev_klabels,
								    NULL,
									dev_kseed_sita_n,
									dev_kseed_m_n,
									dev_kseed_L_n,
									dev_kseed_X_n,
									dev_kseed_Y_n);
		}
	}
#if 1
	////////////////////////////////////////////////////////////////////////////////////////
	cudaStatus=hipMemcpy(host_klabels,dev_klabels,m_width*m_height*sizeof(int),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsl,dev_kseedsl, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsa,dev_kseedsa, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsb,dev_kseedsb, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsx,dev_kseedsx, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsy,dev_kseedsy, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);
#endif
	///////////////////////////////////////////////////////////////////////////////////////
	{
		hipFree(dev_kseedsl);
		hipFree(dev_kseedsa);
		hipFree(dev_kseedsb);
		hipFree(dev_kseedsx);
		hipFree(dev_kseedsy);
		hipFree(dev_klabels);
		hipFree(dev_m_lvec);
		hipFree(dev_m_avec);
		hipFree(dev_m_bvec);
		hipFree(dev_distvec);
		////////////////////////
		hipFree(dev_sita_n);
		hipFree(dev_m_n);
		hipFree(dev_L_n);
		hipFree(dev_X_n);
		hipFree(dev_Y_n);
		{
			//////////////////////////
			hipFree(dev_kseed_sita_n) ;
			hipFree(dev_kseed_m_n);
			hipFree(dev_kseed_L_n);
			hipFree(dev_kseed_X_n);
			hipFree(dev_kseed_Y_n);
		}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	assert(cudaDeInit_CUI()==true);
}
/*------------------------------------------------------------------------------------------*/
/**
*cuda方法实行超像素算法。
*
*
*@param    alpha  角度参数alpha
*@param    betta  模值betta
*@param    gama   亮度gama
*@param    fai    XY距离
*@param    kseedsSize  种子个数
*@param    m_width   图像宽度
*@param    m_height  图像高度

*@param	   klabels  图像图块的标定图像
*@param	   STEP     未使用
*@param	   edgemag  未使用
*@param	   M        未使用密度因素
*
*
*@note
*利用GPU实现超像素算法
*小图1.8S  大图7.2S
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_ThetaMLXY_gpu_simplify(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,   
	const int m_height,
	double*     host_sita_n,
	double*     host_m_n,
	double*     host_L_n,
	double*     host_X_n,
	double*     host_Y_n,
	double*     host_kseedsTheta,
	double*     host_kseedsM,
	double*     host_kseedsL,
	double*     host_kseedsX,
	double*     host_kseedsY,
	int*		host_klabels,
	const int			STEP)
{
	hipError_t cudaStatus;

	double*  dev_kseed_sita_n;
	double*  dev_kseed_m_n;
	double*  dev_kseed_L_n;
	double*  dev_kseed_X_n;
	double*  dev_kseed_Y_n;	
	double*	 dev_sita_n;
	double*	 dev_m_n;
	double*	 dev_L_n;
	double*	 dev_X_n;
	double*	 dev_Y_n;
	int*     dev_klabels;
	double*  dev_distvec;
	assert(cudaInit_CUI()==true);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_klabels,m_width*m_height * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_distvec, m_width*m_height* sizeof(double));

		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize* sizeof(double));

		cudaStatus = hipMalloc((void**)&dev_sita_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_L_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_X_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_Y_n, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMemcpy(dev_klabels, host_klabels, m_width*m_height * sizeof(int), hipMemcpyHostToDevice);
		
		cudaStatus = hipMemcpy(dev_kseed_sita_n, host_kseedsTheta, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseed_m_n, host_kseedsM, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseed_L_n, host_kseedsL, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseed_X_n, host_kseedsX, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseed_Y_n, host_kseedsY, kseedsSize * sizeof(double), hipMemcpyHostToDevice);

		cudaStatus = hipMemcpy(dev_sita_n, host_sita_n, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_n, host_m_n,m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_L_n, host_L_n,m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_X_n, host_X_n,m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_Y_n, host_Y_n,m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
	hipEvent_t start,stop;
	float costtime_ms=0;
	float costtime_us=0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	for( int itr = 0; itr <10; itr++ )
	{
		{
			cudaStatus=hipEventRecord(start,0);
			SetMatrix_Efficient<<<16,16*16>>>(dev_distvec,m_width,m_height,1.7976931348623158e+308);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+threadsPerBlock.x-1)/threadsPerBlock.x,(m_height+threadsPerBlock.y-1)/threadsPerBlock.y);
#if 1
			SetMinimumOnThetaMLXY_9Seeds<<<numBlock,threadsPerBlock>>>(			
							kseedsSize,
							alpha,
							betta,
							gama,
							fai,
							m_width,
							m_height,			
							dev_sita_n,
							dev_m_n,
							dev_L_n,
							dev_X_n,
							dev_Y_n,				
							dev_kseed_sita_n,
							dev_kseed_m_n,
							dev_kseed_L_n,
							dev_kseed_X_n,
							dev_kseed_Y_n,
							dev_klabels,
							dev_distvec);

			cudaGetLastError_Sync_CUI();

			StatisticsThetaMLXY_Seed_collect_CPU(
							kseedsSize,
							m_width,
							m_height,
							host_sita_n,
							host_m_n,
							host_L_n,
							host_X_n,
							host_Y_n,	
							dev_klabels,
    						dev_kseed_sita_n,
							dev_kseed_m_n,
							dev_kseed_L_n,
							dev_kseed_X_n,
							dev_kseed_Y_n);
#endif
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
}	
		hipMemcpy(host_klabels,dev_klabels,m_width*m_height*sizeof(int),hipMemcpyDeviceToHost);		
		hipMemcpy(host_kseedsTheta,dev_kseed_sita_n,kseedsSize * sizeof(double),hipMemcpyDeviceToHost);		
		hipMemcpy(host_kseedsM,dev_kseed_m_n,kseedsSize * sizeof(double),hipMemcpyDeviceToHost);		
		hipMemcpy(host_kseedsL,dev_kseed_L_n,kseedsSize * sizeof(double),hipMemcpyDeviceToHost);		
		hipMemcpy(host_kseedsX,dev_kseed_X_n,kseedsSize * sizeof(double),hipMemcpyDeviceToHost);		
		hipMemcpy(host_kseedsY,dev_kseed_Y_n,kseedsSize * sizeof(double),hipMemcpyDeviceToHost);

	    hipFree(dev_sita_n);
		hipFree(dev_m_n);
		hipFree(dev_L_n);
		hipFree(dev_X_n);
		hipFree(dev_Y_n);
		hipFree(dev_kseed_sita_n) ;
		hipFree(dev_kseed_m_n);
		hipFree(dev_kseed_L_n);
		hipFree(dev_kseed_X_n);
		hipFree(dev_kseed_Y_n);
		hipFree(dev_distvec);
		hipFree(dev_klabels);
	assert(cudaDeInit_CUI()==true);
}
/*------------------------------------------------------------------------------------------*/
/**
*绘制轮廓核函数
*
*@param		[in,out]	ubuff  图像数据
*@param		[in]		labels 图像labels
*@param					width  图像宽度
*@param					height 图像高度
*@param					color_in   颜色内边界颜色
*@param					color_out  颜色外边界颜色
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void DrawContoursAroundSegments_thread(
	unsigned int*			ubuff,
	int*					labels,
	const int				width,
	const int				height,
	const unsigned int		color_in,
	const unsigned int      color_out) 
{
	int k =blockIdx.x*blockDim.x+threadIdx.x;
	int j =blockIdx.y*blockDim.y+threadIdx.y;
	int mainindex = j*width + k;
	const int dx8[8] = {-1, -1,  0,  1, 1, 1, 0, -1};
	const int dy8[8] = { 0, -1, -1, -1, 0, 1, 1,  1};
	////////////////////////////////////////////////
		int np(0);
		if (k>=0&&k<width&&j>=0&&j<height){
				for(register int i = 0; i < 8; i++ ){
					int x = k + dx8[i];
					int y = j + dy8[i];
					if( (x >= 0 && x < width) && (y >= 0 && y < height) ){
								int index = y*width + x;
								//if( false == istaken[index] )//comment this to obtain internal contours
								{
									if( labels[mainindex] != labels[index] ){
										np++;
										//ubuff[index] =0xff000000; //2pixel外线
										ubuff[index] =color_out; //2pixel外线
									}
								}
					}
				}
				if( np > 1 ){
						ubuff[mainindex] =color_in;//2pixel内线
				}
		}		
}
/*------------------------------------------------------------------------------------------*/
/**
*利用GPU方法绘制轮廓到图像上
*@param	  [in,out] ubuff     图像
*@param	  [in] labels    labels矩阵
*@param	   width     图像宽度
*@param	   height    图像高度
*@param	   color_in  轮廓内边界颜色
*@param    color_out 轮廓外边界颜色
*
*/
/*------------------------------------------------------------------------------------------*/
void DrawContoursAroundSegments_gpu(
	unsigned int*			ubuff,
	int*					labels,
	const int				width,
	const int				height,
	const unsigned int	color_in,
	const unsigned int  color_out)
{
	unsigned int* ubuff_dev;
	int*  labels_dev;
	hipError_t cudaStatus;
	assert(cudaInit_CUI()==true);
	hipStream_t stream;//流的结构体

	cudaStatus=hipStreamCreate(&stream);	
	cudaStatus=hipMalloc((void**)&ubuff_dev, width*height*sizeof(unsigned int));
	cudaStatus=hipMalloc((void**)&labels_dev,width*height*sizeof(int));

	cudaStatus = hipMemcpy(ubuff_dev,ubuff, width*height*sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(labels_dev,labels,width*height*sizeof(int), hipMemcpyHostToDevice);
	/////////////////////////////////////////////////////////////////////////////////////////
	dim3    threadsPerBlock(16,16);
	dim3    numBlock((width+threadsPerBlock.x-1)/threadsPerBlock.x,(height+threadsPerBlock.y-1)/threadsPerBlock.y);
	DrawContoursAroundSegments_thread<<<numBlock,threadsPerBlock>>>(ubuff_dev,
											labels_dev,
											width,
											height,
											color_in,
											color_out);
	cudaGetLastError_Sync_CUI();
	hipMemcpy(ubuff,ubuff_dev,width*height*sizeof(unsigned int),hipMemcpyDeviceToHost);

	cudaStatus=hipStreamDestroy(stream);
	hipFree(ubuff_dev);
	hipFree(labels_dev);
	/////////////////////////////////////////////////////////////////////////////////////////
	assert(cudaDeInit_CUI()==true);
}
/*------------------------------------------------------------------------------------------*/
/**
*Normal  0x00 00 00 00 
*MASK    0xff ff ff ff
*设置超像素的屏蔽数据\n
*屏蔽掉轮廓
*@param
*@param	[out]	dev_mask      被屏蔽的轮廓
*@param	[in]	dev_labels    labels矩阵
*@param		width         图像宽度
*@param		height        图像高度
*@param     Pixel         超像素
*/
/*------------------------------------------------------------------------------------------*/
void InitMaskTemplate(
	unsigned int*			dev_mask,
	int*					dev_labels,
	const int				width,
	const int				height,
	const int                Pixel)
{
	int color_in;
	int color_out;
	if (Pixel==2){
		color_in=0xffffffff;
		color_out=0x00000000;
	}else if(Pixel==4){		
		color_in=0xffffffff;
		color_out=0xffffffff;
	}
	
	hipMemset(dev_mask,0,sizeof(unsigned int)*width*height);
	dim3    threadsPerBlock(16,16);
	dim3    numBlock((width+threadsPerBlock.x-1)/threadsPerBlock.x,(height+threadsPerBlock.y-1)/threadsPerBlock.y);
	DrawContoursAroundSegments_thread<<<numBlock,threadsPerBlock>>>(dev_mask,
		dev_labels,
		width,
		height,
		color_in,
		color_out);
	cudaGetLastError_Sync_CUI();
}
/*------------------------------------------------------------------------------------------*/
/**
*
*利用GPU方法获取种子
*.cu本地调用
*
*@param			 kseedsSize   种子个数
*@param			 m_width      图像宽度
*@param			 m_height     图像高度
*@param 	[in]	 host_m_lvec  种子L分量
*@param 	[in]	 host_m_avec  种子A分量
*@param 	[in]	 host_m_bvec  种子B分量
*@param 	[out]	 host_kseedsl  种子L分量
*@param 	[out]	 host_kseedsa  种子A分量
*@param 	[out]	 host_kseedsb  种子B分量
*@param 	[out]	 host_kseedsx  种子X分量
*@param 	[out]	 host_kseedsy  种子Y分量
*@param 	[in]	 host_klabels  图像图块的标定图像
*
*/
/*------------------------------------------------------------------------------------------*/
void  GetSeedsLabxy_gpu(
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				host_m_lvec,
	double*				host_m_avec,
	double*				host_m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels)
{
	hipEvent_t start,stop;
	float costtime_ms=0;
	float costtime_us=0;
	hipError_t cudaStatus;
	unsigned int*       mask_img;
	double*				dev_kseedsl;
	double*				dev_kseedsa;
	double*				dev_kseedsb;
	double*				dev_kseedsx;
	double*				dev_kseedsy;
	int*				dev_klabels;
	double*			dev_m_lvec;
	double*			dev_m_avec;
	double*			dev_m_bvec;
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		cudaStatus=hipMalloc((void**)&mask_img,sizeof(unsigned int)*m_width*m_height);
		cudaStatus = hipMalloc((void**)&dev_kseedsl, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsb, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsx, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsy, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_klabels,m_width*m_height * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_m_lvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_avec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_bvec,m_width*m_height * sizeof(double));
#if 1
		cudaStatus = hipMemset(dev_kseedsl,0, kseedsSize * sizeof(double));
		cudaStatus = hipMemset(dev_kseedsa,0, kseedsSize * sizeof(double));
		cudaStatus = hipMemset(dev_kseedsb,0, kseedsSize * sizeof(double));
		cudaStatus = hipMemset(dev_kseedsx,0, kseedsSize * sizeof(double));
		cudaStatus = hipMemset(dev_kseedsy,0, kseedsSize * sizeof(double));
#endif
		cudaStatus = hipMemcpy(dev_klabels, host_klabels, m_width*m_height * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_lvec, host_m_lvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_avec, host_m_avec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_bvec, host_m_bvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
	}
	
	double*			dev_sigmal;
	double*			dev_sigmaa;
	double*			dev_sigmab;
	double*			dev_sigmax;
	double*			dev_sigmay;
	double*			dev_clustersize;
	double*				dev_kseed_sita_n;
	double*				dev_kseed_m_n;
	double*				dev_kseed_L_n;
	double*				dev_kseed_X_n;
	double*			    dev_kseed_Y_n;
	{
		cudaStatus = hipMalloc((void**)&dev_sigmal, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmaa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmab, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmax, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmay, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_clustersize, kseedsSize * sizeof(double));
		cudaStatus=hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
		cudaGetLastError_Sync_CUI();
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize * sizeof(double));
	}
		/*----------------------------------------------------------------------------------*/
			cudaStatus =hipMemset(dev_sigmal,0,kseedsSize*sizeof(double));			
			cudaStatus =hipMemset(dev_sigmaa,0,kseedsSize*sizeof(double));			
			cudaStatus =hipMemset(dev_sigmab,0,kseedsSize*sizeof(double));			
			cudaStatus =hipMemset(dev_sigmax,0,kseedsSize*sizeof(double));			
			cudaStatus =hipMemset(dev_sigmay,0,kseedsSize*sizeof(double));			
			cudaStatus =hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));			
		/*----------------------------------------------------------------------------------*/

			InitMaskTemplate(mask_img,dev_klabels,m_width,m_height,4);

			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+15)/threadsPerBlock.x,(m_height+15)/threadsPerBlock.y);
			cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
#if 0
			StatisticsLABXY_Seed_collect_gpu_search_AllImg<<<(kseedsSize+511)/512,512>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
#else
			StatisticsLABXY_Seed_collect_CPU(
				kseedsSize,
				m_width,
				m_height,
				host_m_lvec,
				host_m_avec,
				host_m_bvec,
				dev_klabels,
				mask_img,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
#endif
			
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;			
			cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
			StatisticsLABXY_Seed_calculate<<<1,512>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize,
				dev_kseedsl,
				dev_kseedsa,
				dev_kseedsb,
				dev_kseedsx,
				dev_kseedsy,
				dev_kseed_sita_n,
				dev_kseed_m_n,
				dev_kseed_L_n,
				dev_kseed_X_n,
				dev_kseed_Y_n);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
#if 1
		////////////////////////////////////////////////////////////////////////////////////////
		cudaStatus=hipMemcpy(host_klabels,dev_klabels,m_width*m_height*sizeof(int),hipMemcpyDeviceToHost);	
		cudaStatus=hipMemcpy(host_kseedsl,dev_kseedsl, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
		cudaStatus=hipMemcpy(host_kseedsa,dev_kseedsa, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
		cudaStatus=hipMemcpy(host_kseedsb,dev_kseedsb, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
		cudaStatus=hipMemcpy(host_kseedsx,dev_kseedsx, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
		cudaStatus=hipMemcpy(host_kseedsy,dev_kseedsy, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);
#endif
		{
			hipEventDestroy(start);
			hipEventDestroy(stop);
			hipFree(mask_img);
			hipFree(dev_kseedsl);
			hipFree(dev_kseedsa);
			hipFree(dev_kseedsb);
			hipFree(dev_kseedsx);
			hipFree(dev_kseedsy);
			hipFree(dev_klabels);
			hipFree(dev_m_lvec);
			hipFree(dev_m_avec);
			hipFree(dev_m_bvec);
		/*	cudaFree(dev_distvec);*/
			////////////////////////
			/*cudaFree(dev_sita_n);
			cudaFree(dev_m_n);
			cudaFree(dev_L_n);
			cudaFree(dev_X_n);
			cudaFree(dev_Y_n);*/
			{
				hipFree(dev_sigmal);
				hipFree(dev_sigmaa);
				hipFree(dev_sigmab);
				hipFree(dev_sigmax);
				hipFree(dev_sigmay);
				hipFree(dev_clustersize);
				//////////////////////////
				hipFree(dev_kseed_sita_n) ;
				hipFree(dev_kseed_m_n);
				hipFree(dev_kseed_L_n);
				hipFree(dev_kseed_X_n);
				hipFree(dev_kseed_Y_n);
			}
		}
}
/*------------------------------------------------------------------------------------------*/
/**
*
*利用GPU方法获取种子
*.cu本地调用
*
*@param			 kseedsSize   种子个数
*@param			 m_width      图像宽度
*@param			 m_height     图像高度
*@param 	[in]	 host_m_lvec  种子L分量
*@param 	[in]	 host_m_avec  种子A分量
*@param 	[in]	 host_m_bvec  种子B分量
*@param 	[out]	 host_kseedsl  种子L分量
*@param 	[out]	 host_kseedsa  种子A分量
*@param 	[out]	 host_kseedsb  种子B分量
*@param 	[out]	 host_kseedsx  种子X分量
*@param 	[out]	 host_kseedsy  种子Y分量
*@param 	[in]	 host_klabels  图像图块的标定图像
*
*/
/*------------------------------------------------------------------------------------------*/
void  GetSeedsThetaMLXY_gpu(
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*     host_sita_n,
	double*     host_m_n,
	double*     host_L_n,
	double*     host_X_n,
	double*     host_Y_n,
	double*     host_kseedsTheta,
	double*     host_kseedsM,
	double*     host_kseedsL,
	double*     host_kseedsX,
	double*     host_kseedsY,
	int*		host_klabels)
{
	/*		cudaStatus = cudaMalloc((void**)&dev_kseed_sita_n, kseedsSize* sizeof(double));
		cudaStatus = cudaMalloc((void**)&dev_kseed_m_n, kseedsSize* sizeof(double));
		cudaStatus = cudaMalloc((void**)&dev_kseed_L_n, kseedsSize* sizeof(double));
		cudaStatus = cudaMalloc((void**)&dev_kseed_X_n, kseedsSize* sizeof(double));
		cudaStatus = cudaMalloc((void**)&dev_kseed_Y_n, kseedsSize* sizeof(double));*/

}
/*------------------------------------------------------------------------------------------*/
/**
*计算图块的连接矩阵\n
*.cu本地调用
*
*@param	[in]	host_labels      labels矩阵
*@param		    NumLabels       label个数
*@param	[out]	host_Matrix_E   连接矩阵
*@param		    width         图像宽度
*@param		    height        图像高度
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void Nighbour_E_matrix_thread(
	int*			labels,
	const  int		NumLabels,
	UINT32 *		Matrix_E,
	const int		width,
	const int		height)
{
#if 0
int k =blockIdx.x*blockDim.x+threadIdx.x;
	int j =blockIdx.y*blockDim.y+threadIdx.y;
	int mainindex = j*width + k;
	const int dx8[8] = {-1, -1,  0,  1, 1, 1, 0, -1};
	const int dy8[8] = { 0, -1, -1, -1, 0, 1, 1,  1};
	////////////////////////////////////////////////
	int cind(0);
//	static int DoCount=0;
	for (register int ii=0;ii<NumLabels;ii++) {
		Matrix_E[ii*NumLabels+ii]=0;
	}

	int np(0);
	for(register  int i = 0; i < 8; i++ ){
		int x = k + dx8[i];
		int y = j + dy8[i];
		if( (x >= 0 && x < width) && (y >= 0 && y <height) ){
			int index = y*width + x;
			if( labels[mainindex] != labels[index] ){
				UINT32  matrix_x=labels[index];
				UINT32  matrix_y=labels[mainindex];
				Matrix_E[matrix_x*NumLabels+matrix_y]=++Matrix_E[matrix_y*NumLabels+matrix_x];
				if (Matrix_E[matrix_x*NumLabels+matrix_y]>999*2){
					Matrix_E[matrix_x*NumLabels+matrix_y]=Matrix_E[matrix_y*NumLabels+matrix_x]=999*2;
				}
			}
		}
	}
	mainindex++;
	for(register  int i = 0; i <NumLabels; i++ ){
		for(register  int j = 0; j <NumLabels; j++ ){			
			Matrix_E[i*NumLabels+j]=Matrix_E[i*NumLabels+j]>0?1:0;
		}
	}
	{
		double  threshold;		//计算超像素面积
		threshold=4*sqrt((float)width*height/NumLabels);	//周长
		threshold=threshold/8; //8分之一周长

		for(register  int i = 0; i <NumLabels; i++ ){
			for(register  int j = 0; j <NumLabels; j++ ){
				if (Matrix_E[i*NumLabels+j]>0){
					Matrix_E[i*NumLabels+j]=1;     
				}else{
					Matrix_E[i*NumLabels+j]=0;    
				}
			}
		}
	}
#endif
	int k =blockIdx.x*blockDim.x+threadIdx.x;
	int j =blockIdx.y*blockDim.y+threadIdx.y;
	int mainindex = j*width + k;
	const int dx8[8] = {-1, -1,  0,  1, 1, 1, 0, -1};
	const int dy8[8] = { 0, -1, -1, -1, 0, 1, 1,  1};
	/*********************************************************************************************/
	if( (k >= 0 && k < width) && (j >= 0 && j <height) ){
			for(register  int i = 0; i < 8; i++ ){
					int x = k + dx8[i];
					int y = j + dy8[i];
					if( (x >= 0 && x < width) && (y >= 0 && y <height) ){
						int index = y*width + x;
						if( labels[mainindex] != labels[index] ){
							UINT32  matrix_x=labels[index];
							UINT32  matrix_y=labels[mainindex];
							Matrix_E[matrix_x*NumLabels+matrix_y]=Matrix_E[matrix_y*NumLabels+matrix_x]=1;
							//if (matrix_x==matrix_y) assert(Matrix_E[matrix_x*NumLabels+matrix_y]==0);
						}
					}
				}
	}
	/*********************************************************************************************/
}
/*------------------------------------------------------------------------------------------*/
/**
*计算图块的连接矩阵\n
*C函数调用
*
*@param	[in]	host_labels      labels矩阵
*@param		    NumLabels       label个数
*@param	[out]	host_Matrix_E   连接矩阵
*@param		    width         图像宽度
*@param		    height        图像高度
*
*/
/*------------------------------------------------------------------------------------------*/
void Get_Nighbour_E_matrix_gpu(
	int*			host_labels,
	const  int		NumLabels,
	UINT32 *		host_Matrix_E,
	const int		width,
	const int		height)
{
	/*---------------------------*/
	hipError_t cudaStatus;
	int*			dev_labels;
	UINT32 *		dev_Matrix_E;
	/*---------------------------*/
	cudaStatus = hipMalloc((void**)&dev_labels, width*height * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_Matrix_E, NumLabels*NumLabels* sizeof(UINT32));
	cudaStatus=hipMemcpy(dev_labels,host_labels,width*height*sizeof(int),hipMemcpyHostToDevice);
	cudaStatus=hipMemcpy(dev_Matrix_E,host_Matrix_E,NumLabels*NumLabels*sizeof(UINT32),hipMemcpyHostToDevice);
	/*--------------------------------------------*/
	dim3    threadsPerBlock(16,16);
	dim3    numBlock((width+threadsPerBlock.x-1)/threadsPerBlock.x,(height+threadsPerBlock.y-1)/threadsPerBlock.y);
	 Nighbour_E_matrix_thread<<<numBlock,threadsPerBlock>>>(dev_labels,
		NumLabels,
		dev_Matrix_E,
		width,
		height);
	cudaGetLastError_Sync_CUI();
	/*--------------------------------------------*/
	cudaStatus=hipMemcpy(host_Matrix_E,dev_Matrix_E,NumLabels*NumLabels*sizeof(UINT32),hipMemcpyDeviceToHost);
	hipFree(dev_labels);
	hipFree(dev_Matrix_E);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
int GetThreadIdSelfwinlinux(void)
{
		
		  int thread_id=0;
#if _DEBUG
		  int device_count;
		  hipGetDeviceCount(&device_count);
#endif

		
#if _MSC_VER
		  thread_id =::GetCurrentThreadId();
#elif linux||__linux||__linux__||__GNUC__
		  //thread_id=(struct pthread_fake *)pthread_self()->tid;
		  thread_id=syscall( __NR_gettid );
#else
		  thread_id=-1;
		  assert(0);
#endif

	return thread_id;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
//int GetCudaNum(){
//
//}

/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/